#include <diff_sim/global_diff_sim_manager.h>
#include <diff_sim/diff_dof_reporter.h>
#include <diff_sim/diff_parm_reporter.h>
#include <linear_system/global_linear_system.h>
#include <sim_engine.h>
#include <utils/offset_count_collection.h>
#include <sim_engine.h>
#include <kernel_cout.h>

namespace uipc::backend
{
template <>
class backend::SimSystemCreator<cuda::GlobalDiffSimManager>
{
  public:
    static U<cuda::GlobalDiffSimManager> create(SimEngine& engine)
    {
        auto scene = dynamic_cast<SimEngine&>(engine).world().scene();
        auto diff_sim_enable_attr = scene.config().find<IndexT>("diff_sim/enable");

        if(!diff_sim_enable_attr->view()[0])
        {
            return nullptr;
        }
        return uipc::make_unique<cuda::GlobalDiffSimManager>(engine);
    }
};
}  // namespace uipc::backend

namespace uipc::backend::cuda
{
namespace detail
{
    void build_coo_matrix(muda::LinearSystemContext&           ctx,
                          muda::DeviceCOOMatrix<Float>&        total_coo,
                          muda::DeviceTripletMatrix<Float, 1>& total_triplet,
                          muda::DeviceTripletMatrix<Float, 1>& local_triplet)
    {
        using namespace muda;

        // 1) reshape the total_coo and total_triplet
        auto M = local_triplet.rows();
        auto N = local_triplet.cols();
        total_coo.reshape(M, N);
        total_triplet.reshape(M, N);

        // 2) append the local_triplet to the total_triplet
        //  2.1) resize copy the total_coo to total_triplet
        auto new_triplet_count = total_coo.non_zeros() + local_triplet.triplet_count();
        total_triplet.resize_triplets(new_triplet_count);
        auto total_triplet_view = total_triplet.view();
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(total_coo.non_zeros(),
                   [total_coo     = total_coo.cviewer().name("total_coo"),
                    total_triplet = total_triplet_view
                                        .subview(0, total_coo.non_zeros())  // front
                                        .viewer()
                                        .name("total_triplet")] __device__(int I) mutable
                   {
                       auto&& [i, j, V] = total_coo(I);
                       total_triplet(I).write(i, j, V);
                   });
        //  2.2) append the local_triplet to the total_triplet
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(local_triplet.triplet_count(),
                   [local_triplet = local_triplet.cviewer().name("local_triplet"),
                    total_triplet = total_triplet_view
                                        .subview(total_coo.non_zeros(),
                                                 local_triplet.triplet_count())  // back
                                        .viewer()
                                        .name("total_triplet")] __device__(int I) mutable
                   {
                       auto&& [i, j, V] = local_triplet(I);
                       total_triplet(I).write(i, j, V);
                       // cout << "i: " << i << ", j: " << j << ", V: " << V << "\n";
                   })
            .wait();

        // 3) convert the total_triplet to total_coo
        ctx.convert(total_triplet, total_coo);
    }

    void copy_to_host(const muda::DeviceCOOMatrix<Float>& total_coo,
                      GlobalDiffSimManager::SparseCOO&    host_coo)
    {
        // copy row_inides, col_indices, values to host_coo

        host_coo.row_indices.resize(total_coo.row_indices().size());
        total_coo.row_indices().copy_to(host_coo.row_indices.data());

        host_coo.col_indices.resize(total_coo.col_indices().size());
        total_coo.col_indices().copy_to(host_coo.col_indices.data());

        host_coo.values.resize(total_coo.values().size());
        total_coo.values().copy_to(host_coo.values.data());

        host_coo.shape = {total_coo.rows(), total_coo.cols()};
    }
}  // namespace detail
// namespace detail
}  // namespace uipc::backend::cuda

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalDiffSimManager);

void GlobalDiffSimManager::do_build()
{
    m_impl.global_linear_system = &require<GlobalLinearSystem>();
    m_impl.sim_engine           = &engine();

    on_write_scene([&] { m_impl.write_scene(world()); });
}

muda::LinearSystemContext& GlobalDiffSimManager::Impl::ctx()
{
    return global_linear_system->m_impl.ctx;
}

void GlobalDiffSimManager::Impl::init(WorldVisitor& world)
{
    auto& diff_sim   = world.scene().diff_sim();
    auto  parm_view  = diff_sim.parameters().view();
    total_parm_count = parm_view.size();
    dof_offsets.reserve(1024);
    dof_counts.reserve(1024);
    total_coo_pGpP.reshape(0, 0);
    total_coo_H.reshape(0, 0);

    // 1) Copy the parameters to the device
    parameters.resize(total_parm_count);
    parameters.view().copy_from(parm_view.data());


    // 2) Init the diff_parm_reporters
    {
        auto diff_parm_reporter_view = diff_parm_reporters.view();
        for(auto&& [i, R] : enumerate(diff_parm_reporter_view))
        {
            R->m_index = i;
        }

        diff_parm_triplet_offset_count.resize(diff_parm_reporter_view.size());
    }

    // 3) Init the diff_dof_reporters
    {
        auto diff_dof_reporter_view = diff_dof_reporters.view();
        for(auto&& [i, R] : enumerate(diff_dof_reporter_view))
        {
            R->m_index = i;
        }

        diff_dof_triplet_offset_count.resize(diff_dof_reporter_view.size());
    }
}

void GlobalDiffSimManager::Impl::update()
{
    // Waiting for later version merging
}

void GlobalDiffSimManager::Impl::assemble()
{
    // Waiting for later version merging
}

void GlobalDiffSimManager::Impl::write_scene(WorldVisitor& world)
{
    // Waiting for later version merging
}

void GlobalDiffSimManager::init()
{
    m_impl.init(world());
}

void GlobalDiffSimManager::assemble()
{
    m_impl.assemble();
}

void GlobalDiffSimManager::update()
{
    m_impl.update();
}

void GlobalDiffSimManager::add_reporter(DiffDofReporter* subsystem)
{
    UIPC_ASSERT(subsystem != nullptr, "subsystem is nullptr");
    m_impl.diff_dof_reporters.register_subsystem(*subsystem);
}

void GlobalDiffSimManager::add_reporter(DiffParmReporter* subsystem)
{
    UIPC_ASSERT(subsystem != nullptr, "subsystem is nullptr");
    m_impl.diff_parm_reporters.register_subsystem(*subsystem);
}

muda::TripletMatrixView<Float, 1> GlobalDiffSimManager::DiffParmInfo::pGpP() const
{
    auto offset = m_impl->diff_parm_triplet_offset_count.offsets()[m_index];
    auto count  = m_impl->diff_parm_triplet_offset_count.counts()[m_index];
    return m_impl->local_triplet_pGpP.view().subview(offset, count);
}

muda::TripletMatrixView<Float, 1> GlobalDiffSimManager::DiffDofInfo::H() const
{
    auto offset = m_impl->diff_dof_triplet_offset_count.offsets()[m_index];
    auto count  = m_impl->diff_dof_triplet_offset_count.counts()[m_index];
    return m_impl->local_triplet_H.view().subview(offset, count);
}

SizeT GlobalDiffSimManager::BaseInfo::frame() const
{
    return m_impl->sim_engine->frame();
}

IndexT GlobalDiffSimManager::BaseInfo::dof_offset(SizeT frame) const
{
    return m_impl->dof_offsets[frame - 1];  // we record from the frame 1
}

IndexT GlobalDiffSimManager::BaseInfo::dof_count(SizeT frame) const
{
    return m_impl->dof_counts[frame - 1];  // we record from the frame 1
}

diff_sim::SparseCOOView GlobalDiffSimManager::SparseCOO::view() const
{
    return diff_sim::SparseCOOView{row_indices, col_indices, values, shape};
}

muda::CBufferView<Float> GlobalDiffSimManager::DiffParmUpdateInfo::parameters() const noexcept
{
    return m_impl->parameters.view();
}
}  // namespace uipc::backend::cuda