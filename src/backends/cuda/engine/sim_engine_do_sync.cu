#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <muda/launch/launch.h>

namespace uipc::backend::cuda
{
void SimEngine::do_sync()
{
    try
    {
        // Sync the device
        muda::wait_device();
    }
    catch(const SimEngineException& e)
    {
        log::error("SimEngine Sync Error: {}", e.what());
        status().push_back(core::EngineStatus::error(e.what()));
    }
}
}  // namespace uipc::backend::cuda
