#include <sim_engine.h>

namespace uipc::backend::cuda
{
void SimEngine::do_retrieve()
{
    try
    {
        event_write_scene();
    }
    catch(const SimEngineException& e)
    {
        logger::error("SimEngine Retrieve Error: {}", e.what());
        status().push_back(core::EngineStatus::error(e.what()));
    }
}

}  // namespace uipc::backend::cuda
