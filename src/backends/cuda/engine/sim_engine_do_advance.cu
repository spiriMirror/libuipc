#include "hip/hip_runtime.h"
#include <sim_engine.h>
#include <uipc/common/range.h>
#include <global_geometry/global_vertex_manager.h>
#include <global_geometry/global_simplicial_surface_manager.h>
#include <dytopo_effect_system/global_dytopo_effect_manager.h>
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_trajectory_filter.h>
#include <line_search/line_searcher.h>
#include <linear_system/global_linear_system.h>
#include <animator/global_animator.h>
#include <diff_sim/global_diff_sim_manager.h>
#include <newton_tolerance/newton_tolerance_manager.h>
#include <time_integrator/time_integrator_manager.h>

namespace uipc::backend::cuda
{
void SimEngine::do_advance()
{
    Float alpha     = 1.0;
    Float ccd_alpha = 1.0;
    Float cfl_alpha = 1.0;

    /***************************************************************************************
    *                                  Function Shortcuts
    ***************************************************************************************/

    auto detect_dcd_candidates = [this]
    {
        if(m_global_trajectory_filter)
        {
            Timer timer{"Detect DCD Candidates"};
            m_global_trajectory_filter->detect(0.0);
            m_global_trajectory_filter->filter_active();
        }
    };

    auto detect_trajectory_candidates = [this](Float alpha)
    {
        if(m_global_trajectory_filter)
        {
            Timer timer{"Detect Trajectory Candidates"};
            m_global_trajectory_filter->detect(alpha);
        }
    };

    auto filter_dcd_candidates = [this]
    {
        if(m_global_trajectory_filter)
        {
            Timer timer{"Filter Contact Candidates"};
            m_global_trajectory_filter->filter_active();
        }
    };

    auto record_friction_candidates = [this]
    {
        if(m_global_trajectory_filter && m_friction_enabled)
        {
            m_global_trajectory_filter->record_friction_candidates();
        }
    };

    auto compute_adaptive_kappa = [this]
    {
        // TODO: now no effect
        if(m_global_contact_manager)
            m_global_contact_manager->compute_adaptive_kappa();
    };

    auto compute_dytopo_effect = [this]
    {
        // compute the dytopo effect gradient and hessian, containing:
        // 1) contact effect from contact pairs
        // 2) other dynamic topo effects, e.g. point picker, vertex stitch ...
        if(m_global_dytopo_effect_manager)
        {
            Timer timer{"Compute DyTopo Effect"};
            m_global_dytopo_effect_manager->compute_dytopo_effect();
        }
    };

    auto cfl_condition = [&cfl_alpha, this](Float alpha)
    {
        if(m_global_contact_manager)
        {
            cfl_alpha = m_global_contact_manager->compute_cfl_condition();
            if(cfl_alpha < alpha)
            {
                spdlog::info("CFL Filter: {} < {}", cfl_alpha, alpha);
                return cfl_alpha;
            }
        }

        return alpha;
    };

    auto filter_toi = [&ccd_alpha, this](Float alpha)
    {
        if(m_global_trajectory_filter)
        {
            Timer timer{"Filter CCD TOI"};
            ccd_alpha = m_global_trajectory_filter->filter_toi(alpha);
            if(ccd_alpha < alpha)
            {
                spdlog::info("CCD Filter: {} < {}", ccd_alpha, alpha);
                return ccd_alpha;
            }
        }

        return alpha;
    };

    auto compute_energy = [this, filter_dcd_candidates](Float alpha) -> Float
    {
        // Step Forward => x = x_0 + alpha * dx
        m_global_vertex_manager->step_forward(alpha);
        m_line_searcher->step_forward(alpha);

        // Update the collision pairs
        filter_dcd_candidates();

        // Compute New Energy => E
        return m_line_searcher->compute_energy(false);
    };

    auto step_animation = [this]()
    {
        if(m_global_animator)
        {
            Timer timer{"Step Animation"};
            m_global_animator->step();
        }
    };

    auto compute_animation_substep_ratio = [this](SizeT newton_iter)
    {
        // compute the ratio to the aim position.
        // dst = prev_position + ratio * (position - prev_position)
        if(m_global_animator)
        {
            m_global_animator->compute_substep_ratio(newton_iter);
            spdlog::info("Animation Substep Ratio: {}", m_global_animator->substep_ratio());
        }
    };

    auto animation_reach_target = [this]()
    {
        if(m_global_animator)
        {
            return m_global_animator->substep_ratio() >= 1.0;
        }
        return true;
    };

    auto update_diff_parm = [this]()
    {
        if(m_global_diff_sim_manager)
        {
            Timer timer{"Update Diff Parm"};
            m_global_diff_sim_manager->update();
        }
    };

    /***************************************************************************************
    *                                  Core Pipeline
    ***************************************************************************************/

    // Abort on exception if the runtime check is enabled for debugging
    constexpr bool AbortOnException = uipc::RUNTIME_CHECK;

    auto pipeline = [&]() noexcept(AbortOnException)
    {
        Timer timer{"Pipeline"};

        ++m_current_frame;

        spdlog::info(R"(>>> Begin Frame: {})", m_current_frame);

        // Rebuild Scene
        {
            Timer timer{"Rebuild Scene"};
            // Trigger the rebuild_scene event, systems register their actions will be called here
            m_state = SimEngineState::RebuildScene;
            {
                event_rebuild_scene();

                // TODO: rebuild the vertex and surface info
                // m_global_vertex_manager->rebuild_vertex_info();
                // m_global_surface_manager->rebuild_surface_info();
            }

            // After the rebuild_scene event, the pending creation or deletion can be solved
            world().scene().solve_pending();

            // Update the diff parms
            update_diff_parm();
        }

        // Simulation:
        {
            Timer timer{"Simulation"};
            // 1. Adaptive Parameter Calculation
            AABB vertex_bounding_box =
                m_global_vertex_manager->compute_vertex_bounding_box();
            detect_dcd_candidates();
            compute_adaptive_kappa();

            // 2. Record Friction Candidates at the beginning of the frame
            record_friction_candidates();
            m_global_vertex_manager->record_prev_positions();

            // 3. Predict Motion => x_tilde = x + v * dt
            m_state = SimEngineState::PredictMotion;
            m_time_integrator_manager->predict_dof();
            step_animation();

            // 4. Nonlinear-Newton Iteration
            Float box_size = vertex_bounding_box.diagonal().norm();
            Float tol      = m_newton_scene_tol * box_size;
            Float res0     = 0.0;
            m_newton_tolerance_manager->pre_newton(m_current_frame);

            auto   newton_max_iter = m_newton_max_iter->view()[0];
            IndexT newton_iter     = 0;
            for(; newton_iter < newton_max_iter; ++newton_iter)
            {
                Timer timer{"Newton Iteration"};

                // 1) Compute animation substep ratio
                compute_animation_substep_ratio(newton_iter);

                // 2) Build Collision Pairs
                if(newton_iter > 0)
                    detect_dcd_candidates();

                // 3) Compute Dynamic Topo Effect Gradient and Hessian => G:Vector3, H:Matrix3x3
                //    Including Contact Effect
                m_state = SimEngineState::ComputeDyTopoEffect;
                compute_dytopo_effect();

                // 4) Solve Global Linear System => dx = A^-1 * b
                m_state = SimEngineState::SolveGlobalLinearSystem;
                {
                    Timer timer{"Solve Global Linear System"};
                    m_global_linear_system->solve();
                }


                // 5) Collect Vertex Displacements Globally
                m_global_vertex_manager->collect_vertex_displacements();


                // 6) Check Termination Condition
                bool converged = false;
                {
                    NewtonToleranceManager::ResultInfo result_info;
                    result_info.frame(m_current_frame);
                    result_info.newton_iter(newton_iter);
                    m_newton_tolerance_manager->check(result_info);

                    converged = result_info.converged();

                    if(m_dump_surface->view()[0])
                    {
                        dump_global_surface(fmt::format(
                            "dump_surface.{}.{}", m_current_frame, newton_iter));
                    }

                    if(converged  // check convergence
                       && ccd_alpha >= m_ccd_tol->view()[0]  // check ccd tolerance
                       && animation_reach_target())  // check animation target
                    {
                        break;
                    }
                }

                // 7) Begin Line Search
                m_state = SimEngineState::LineSearch;
                {
                    Timer timer{"Line Search"};

                    // Reset Alpha
                    alpha = 1.0;

                    // Record Current State x to x_0
                    m_line_searcher->record_start_point();
                    m_global_vertex_manager->record_start_point();
                    detect_trajectory_candidates(alpha);

                    // Compute Current Energy => E_0
                    Float E0 = m_line_searcher->compute_energy(true);  // initial energy
                    // spdlog::info("Initial Energy: {}", E0);

                    // CCD filter
                    alpha = filter_toi(alpha);

                    // CFL Condition
                    alpha = cfl_condition(alpha);

                    // * Step Forward => x = x_0 + alpha * dx
                    // Compute Test Energy => E
                    Float E  = compute_energy(alpha);
                    Float E1 = E;

                    SizeT line_search_iter = 0;
                    while(line_search_iter < m_line_searcher->max_iter())
                    {
                        Timer timer{"Line Search Iteration"};

                        bool energy_decrease = E <= E0;  // Check Energy Decrease

                        // TODO: Inversion Check (Not Implemented Yet)
                        bool no_inversion = true;

                        bool success = energy_decrease && no_inversion;

                        if(success)
                            break;

                        // If not success, then shrink alpha
                        alpha /= 2;
                        E = compute_energy(alpha);

                        line_search_iter++;
                    }

                    if(line_search_iter >= m_line_searcher->max_iter())
                    {
                        spdlog::warn(
                            "Line Search Exits with Max Iteration: {} (Frame={}, Newton={})\n"
                            "E/E0: {}, E1/E0: {}, E0:{}",
                            m_line_searcher->max_iter(),
                            m_current_frame,
                            newton_iter,
                            E / E0,
                            E1 / E0,
                            E0);

                        if(m_strict_mode)
                        {
                            throw SimEngineException("StrictMode: Line Search Exits with Max Iteration");
                        }
                    }
                }
            }

            // 5. Update Velocity => v = (x - x_0) / dt
            m_state = SimEngineState::UpdateVelocity;
            {
                Timer timer{"Update Velocity"};
                m_time_integrator_manager->update_state();
            }


            if(newton_iter > newton_max_iter)
            {
                spdlog::warn("Newton Iteration Exits with Max Iteration: {} (Frame={})",
                             newton_max_iter,
                             m_current_frame);

                if(m_strict_mode)
                {
                    throw SimEngineException("StrictMode: Newton Iteration Exits with Max Iteration");
                }
            }
        }

        spdlog::info("<<< End Frame: {}", m_current_frame);
    };

    try
    {
        pipeline();
        m_last_solved_frame = m_current_frame;
    }
    catch(const SimEngineException& e)
    {
        spdlog::error("Engine Advance Error: {}", e.what());
        status().push_back(core::EngineStatus::error(e.what()));
    }
}
}  // namespace uipc::backend::cuda
