#include "hip/hip_runtime.h"
#include <affine_body/matrix_converter.h>
#include <muda/cub/device/device_merge_sort.h>
#include <muda/cub/device/device_run_length_encode.h>
#include <muda/cub/device/device_scan.h>
#include <muda/cub/device/device_segmented_reduce.h>
#include <muda/cub/device/device_radix_sort.h>
#include <muda/cub/device/device_select.h>
#include <cub/warp/warp_reduce.cuh>
#include <muda/ext/eigen/atomic.h>
#include <uipc/common/timer.h>
#include <algorithm/fast_segmental_reduce.h>
#include <muda/cub/device/device_reduce.h>
#include <muda/cub/device/device_partition.h>

// for encode run length usage
MUDA_GENERIC constexpr bool operator==(const int2& a, const int2& b)
{
    return a.x == b.x && a.y == b.y;
}

namespace uipc::backend::cuda
{
void ABDMatrixConverter::convert(const muda::DeviceTripletMatrix<T, N>& from,
                                 muda::DeviceBCOOMatrix<T, N>&          to)
{
    m_impl.convert(from, to);
}

void ABDMatrixConverter::Impl::convert(const muda::DeviceTripletMatrix<T, N>& from,
                                       muda::DeviceBCOOMatrix<T, N>& to)
{
    to.reshape(from.rows(), from.cols());
    to.resize_triplets(from.triplet_count());


    if(to.triplet_count() == 0)
        return;

    _radix_sort_indices_and_blocks(from, to);
    _make_unique_indices(from, to);
    _make_unique_block_warp_reduction(from, to);
}


void ABDMatrixConverter::Impl::_radix_sort_indices_and_blocks(
    const muda::DeviceTripletMatrix<T, N>& from, muda::DeviceBCOOMatrix<T, N>& to)
{
    using namespace muda;

    auto src_row_indices = from.row_indices();
    auto src_col_indices = from.col_indices();
    auto src_blocks      = from.values();

    loose_resize(ij_hash_input, src_row_indices.size());
    loose_resize(sort_index_input, src_row_indices.size());

    loose_resize(ij_hash, src_row_indices.size());
    loose_resize(sort_index, src_row_indices.size());
    ij_pairs.resize(src_row_indices.size());


    // hash ij
    ParallelFor(256)
        .file_line(__FILE__, __LINE__)
        .apply(src_row_indices.size(),
               [row_indices = src_row_indices.cviewer().name("row_indices"),
                col_indices = src_col_indices.cviewer().name("col_indices"),
                ij_hash     = ij_hash_input.viewer().name("ij_hash"),
                sort_index = sort_index_input.viewer().name("sort_index")] __device__(int i) mutable
               {
                   ij_hash(i) = (static_cast<uint64_t>(row_indices(i)) << 32)
                                + static_cast<uint64_t>(col_indices(i));
                   sort_index(i) = i;
               });

    DeviceRadixSort().SortPairs(ij_hash_input.data(),
                                ij_hash.data(),
                                sort_index_input.data(),
                                sort_index.data(),
                                ij_hash.size());

    // set ij_hash back to row_indices and col_indices

    auto dst_row_indices = to.row_indices();
    auto dst_col_indices = to.col_indices();

    ParallelFor(256)
        .kernel_name("set col row indices")
        .apply(dst_row_indices.size(),
               [ij_hash = ij_hash.viewer().name("ij_hash"),
                ij_pairs = ij_pairs.viewer().name("ij_pairs")] __device__(int i) mutable
               {
                   auto hash      = ij_hash(i);
                   auto row_index = static_cast<int>(hash >> 32);
                   auto col_index = static_cast<int>(hash & 0xFFFFFFFF);
                   ij_pairs(i).x  = row_index;
                   ij_pairs(i).y  = col_index;
               });

    // sort the block values

    {
        Timer timer("set block values");
        loose_resize(blocks_sorted, from.values().size());
        ParallelFor(256)
            .file_line(__FILE__, __LINE__)
            .apply(src_blocks.size(),
                   [src_blocks = src_blocks.cviewer().name("blocks"),
                    sort_index = sort_index.cviewer().name("sort_index"),
                    dst_blocks = blocks_sorted.viewer().name("values")] __device__(int i) mutable
                   { dst_blocks(i) = src_blocks(sort_index(i)); });
    }
}

void ABDMatrixConverter::Impl::_make_unique_indices(const muda::DeviceTripletMatrix<T, N>& from,
                                                    muda::DeviceBCOOMatrix<T, N>& to)
{
    using namespace muda;

    auto row_indices = to.row_indices();
    auto col_indices = to.col_indices();

    loose_resize(unique_ij_pairs, ij_pairs.size());
    loose_resize(unique_counts, ij_pairs.size());


    DeviceRunLengthEncode().Encode(ij_pairs.data(),
                                   unique_ij_pairs.data(),
                                   unique_counts.data(),
                                   count.data(),
                                   ij_pairs.size());

    int h_count = count;

    unique_ij_pairs.resize(h_count);
    unique_counts.resize(h_count);

    offsets.resize(unique_counts.size() + 1);  // +1 for the last offset_end

    DeviceScan().ExclusiveSum(
        unique_counts.data(), offsets.data(), unique_counts.size());


    muda::ParallelFor(256)
        .file_line(__FILE__, __LINE__)
        .apply(unique_counts.size(),
               [unique_ij_pairs = unique_ij_pairs.viewer().name("unique_ij_pairs"),
                row_indices = row_indices.viewer().name("row_indices"),
                col_indices = col_indices.viewer().name("col_indices")] __device__(int i) mutable
               {
                   row_indices(i) = unique_ij_pairs(i).x;
                   col_indices(i) = unique_ij_pairs(i).y;
               });

    to.resize_triplets(h_count);
}

void ABDMatrixConverter::Impl::_make_unique_block_warp_reduction(
    const muda::DeviceTripletMatrix<T, N>& from, muda::DeviceBCOOMatrix<T, N>& to)
{
    using namespace muda;

    loose_resize(sorted_partition_input, ij_pairs.size());
    loose_resize(sorted_partition_output, ij_pairs.size());


    BufferLaunch().fill<int>(sorted_partition_input, 0);

    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(unique_counts.size(),
               [sorted_partition = sorted_partition_input.viewer().name("sorted_partition"),
                unique_counts = unique_counts.viewer().name("unique_counts"),
                offsets = offsets.viewer().name("offsets")] __device__(int i) mutable
               {
                   auto offset = offsets(i);
                   auto count  = unique_counts(i);

                   sorted_partition(offset + count - 1) = 1;
               });

    // scatter
    DeviceScan().ExclusiveSum(sorted_partition_input.data(),
                              sorted_partition_output.data(),
                              sorted_partition_input.size());

    auto blocks = to.values();


    FastSegmentalReduce()
        .file_line(__FILE__, __LINE__)
        .reduce(std::as_const(sorted_partition_output).view(),
                std::as_const(blocks_sorted).view(),
                blocks);
}
}  // namespace uipc::backend::cuda