#include "hip/hip_runtime.h"
#include <collision_detection/filters/lbvh_simplex_trajectory_filter.h>
#include <muda/cub/device/device_select.h>
#include <muda/ext/eigen/log_proxy.h>
#include <sim_engine.h>
#include <kernel_cout.h>
#include <utils/distance/distance_flagged.h>
#include <utils/distance.h>
#include <utils/codim_thickness.h>
#include <utils/simplex_contact_mask_utils.h>
#include <uipc/common/zip.h>
#include <utils/primitive_d_hat.h>

namespace uipc::backend::cuda
{
constexpr bool PrintDebugInfo = false;

REGISTER_SIM_SYSTEM(LBVHSimplexTrajectoryFilter);

void LBVHSimplexTrajectoryFilter::do_build(BuildInfo& info)
{
    auto& config = world().scene().config();
    auto  method = config.find<std::string>("collision_detection/method");
    if(method->view()[0] != "linear_bvh")
    {
        throw SimSystemException("Linear BVH unused");
    }
}

void LBVHSimplexTrajectoryFilter::do_detect(DetectInfo& info)
{
    m_impl.detect(info);
}

void LBVHSimplexTrajectoryFilter::do_filter_active(FilterActiveInfo& info)
{
    m_impl.filter_active(info);
}

void LBVHSimplexTrajectoryFilter::do_filter_toi(FilterTOIInfo& info)
{
    m_impl.filter_toi(info);
}

void LBVHSimplexTrajectoryFilter::Impl::detect(DetectInfo& info)
{
    using namespace muda;

    auto alpha   = info.alpha();
    auto Ps      = info.positions();
    auto dxs     = info.displacements();
    auto codimVs = info.codim_vertices();
    auto Vs      = info.surf_vertices();
    auto Es      = info.surf_edges();
    auto Fs      = info.surf_triangles();

    point_aabbs.resize(Vs.size());
    triangle_aabbs.resize(Fs.size());
    edge_aabbs.resize(Es.size());

    // build AABBs for codim vertices
    if(codimVs.size() > 0)
    {
        codim_point_aabbs.resize(codimVs.size());

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(codimVs.size(),
                   [codimVs = codimVs.viewer().name("codimVs"),
                    Ps      = Ps.viewer().name("Ps"),
                    dxs     = dxs.viewer().name("dxs"),
                    aabbs   = codim_point_aabbs.viewer().name("aabbs"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    d_hats = info.d_hats().viewer().name("d_hats"),
                    alpha  = alpha] __device__(int i) mutable
                   {
                       auto vI = codimVs(i);

                       Float thickness       = thicknesses(vI);
                       Float d_hat_expansion = point_dcd_expansion(d_hats(vI));

                       const auto& pos   = Ps(vI);
                       Vector3     pos_t = pos + dxs(vI) * alpha;

                       AABB aabb;
                       aabb.extend(pos).extend(pos_t);

                       Float expand = d_hat_expansion + thickness;

                       aabb.min().array() -= expand;
                       aabb.max().array() += expand;
                       aabbs(i) = aabb;
                   });
    }

    // build AABBs for surf vertices (including codim vertices)
    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(Vs.size(),
               [Vs          = Vs.viewer().name("V"),
                dxs         = dxs.viewer().name("dx"),
                Ps          = Ps.viewer().name("Ps"),
                aabbs       = point_aabbs.viewer().name("aabbs"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                d_hats      = info.d_hats().viewer().name("d_hats"),
                alpha       = alpha] __device__(int i) mutable
               {
                   auto vI = Vs(i);

                   Float thickness       = thicknesses(vI);
                   Float d_hat_expansion = point_dcd_expansion(d_hats(vI));

                   const auto& pos   = Ps(vI);
                   Vector3     pos_t = pos + dxs(vI) * alpha;

                   AABB aabb;
                   aabb.extend(pos).extend(pos_t);

                   Float expand = d_hat_expansion + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });

    // build AABBs for edges
    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(Es.size(),
               [Es          = Es.viewer().name("E"),
                Ps          = Ps.viewer().name("Ps"),
                aabbs       = edge_aabbs.viewer().name("aabbs"),
                dxs         = dxs.viewer().name("dx"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                d_hats      = info.d_hats().viewer().name("d_hats"),
                alpha       = alpha] __device__(int i) mutable
               {
                   auto eI = Es(i);

                   Float thickness =
                       edge_thickness(thicknesses(eI[0]), thicknesses(eI[1]));
                   Float d_hat_expansion =
                       edge_dcd_expansion(d_hats(eI[0]), d_hats(eI[1]));

                   const auto& pos0   = Ps(eI[0]);
                   const auto& pos1   = Ps(eI[1]);
                   Vector3     pos0_t = pos0 + dxs(eI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(eI[1]) * alpha;

                   Vector3 max = pos0_t;
                   Vector3 min = pos0_t;

                   AABB aabb;

                   aabb.extend(pos0).extend(pos1).extend(pos0_t).extend(pos1_t);

                   Float expand = d_hat_expansion + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });

    // build AABBs for triangles
    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(Fs.size(),
               [Fs          = Fs.viewer().name("F"),
                Ps          = Ps.viewer().name("Ps"),
                aabbs       = triangle_aabbs.viewer().name("aabbs"),
                dxs         = dxs.viewer().name("dx"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                d_hats      = info.d_hats().viewer().name("d_hats"),
                alpha       = alpha] __device__(int i) mutable
               {
                   auto fI = Fs(i);

                   Float thickness = triangle_thickness(thicknesses(fI[0]),
                                                        thicknesses(fI[1]),
                                                        thicknesses(fI[2]));
                   Float d_hat_expansion = triangle_dcd_expansion(
                       d_hats(fI[0]), d_hats(fI[1]), d_hats(fI[2]));

                   const auto& pos0   = Ps(fI[0]);
                   const auto& pos1   = Ps(fI[1]);
                   const auto& pos2   = Ps(fI[2]);
                   Vector3     pos0_t = pos0 + dxs(fI[0]) * alpha;
                   Vector3     pos1_t = pos1 + dxs(fI[1]) * alpha;
                   Vector3     pos2_t = pos2 + dxs(fI[2]) * alpha;

                   AABB aabb;

                   aabb.extend(pos0)
                       .extend(pos1)
                       .extend(pos2)
                       .extend(pos0_t)
                       .extend(pos1_t)
                       .extend(pos2_t);

                   Float expand = d_hat_expansion + thickness;

                   aabb.min().array() -= expand;
                   aabb.max().array() += expand;
                   aabbs(i) = aabb;
               });

    lbvh_E.build(edge_aabbs);
    lbvh_T.build(triangle_aabbs);

    if(codimVs.size() > 0)
    {
        // Use AllP to query CodimP
        {
            lbvh_CodimP.build(codim_point_aabbs);

            muda::KernelLabel label{__FUNCTION__, __FILE__, __LINE__};
            lbvh_CodimP.query(
                point_aabbs,                                  // AllP
                [Vs      = Vs.viewer().name("Vs"),            // AllP
                 codimVs = codimVs.viewer().name("codimVs"),  // CodimP

                 Ps          = Ps.viewer().name("Ps"),
                 dxs         = dxs.viewer().name("dxs"),
                 thicknesses = info.thicknesses().viewer().name("thicknesses"),
                 dimensions  = info.dimensions().viewer().name("dimensions"),
                 contact_element_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                 contact_mask_tabular = info.contact_mask_tabular().viewer().name("contact_mask_tabular"),
                 subscene_element_ids = info.subscene_element_ids().viewer().name("subscene_element_ids"),
                 subscene_mask_tabular = info.subscene_mask_tabular().viewer().name("subscene_mask_tabular"),
                 v2b = info.v2b().viewer().name("v2b"),
                 body_self_collision = info.body_self_collision().viewer().name("body_self_collision"),
                 d_hats = info.d_hats().viewer().name("d_hats"),
                 alpha  = alpha] __device__(IndexT i, IndexT j)
                {
                    const auto& V      = Vs(i);
                    const auto& codimV = codimVs(j);

                    Vector2i cids = {contact_element_ids(V), contact_element_ids(codimV)};
                    Vector2i scids = {subscene_element_ids(V), subscene_element_ids(codimV)};

                    // discard if the contact is disabled
                    if(!allow_PP_contact(subscene_mask_tabular, scids))
                        return false;
                    if(!allow_PP_contact(contact_mask_tabular, cids))
                        return false;

                    bool V_is_codim = dimensions(V) <= 2;  // codim 0D vert and vert from codim 1D edge

                    if(V_is_codim && V >= codimV)  // avoid duplicate CodimP-CodimP pairs
                        return false;

                    auto body_i = v2b(V);
                    auto body_j = v2b(codimV);
                    // skip self-collision for the same body if self collision off
                    if(body_i == body_j && !body_self_collision(body_i))
                        return false;


                    Vector3 P0  = Ps(V);
                    Vector3 dP0 = alpha * dxs(V);

                    Vector3 P1  = Ps(codimV);
                    Vector3 dP1 = alpha * dxs(codimV);

                    Float thickness = PP_thickness(thicknesses(V), thicknesses(codimV));
                    Float d_hat = PP_d_hat(d_hats(V), d_hats(codimV));

                    Float expand = d_hat + thickness;

                    if(!distance::point_point_ccd_broadphase(P0, P1, dP0, dP1, expand))
                        return false;

                    return true;
                },
                candidate_AllP_CodimP_pairs);
        }

        // Use CodimP to query AllE
        {
            muda::KernelLabel label{__FUNCTION__, __FILE__, __LINE__};
            lbvh_E.query(
                codim_point_aabbs,
                [codimVs     = codimVs.viewer().name("Vs"),
                 Es          = Es.viewer().name("Es"),
                 Ps          = Ps.viewer().name("Ps"),
                 dxs         = dxs.viewer().name("dxs"),
                 thicknesses = info.thicknesses().viewer().name("thicknesses"),
                 contact_element_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                 contact_mask_tabular = info.contact_mask_tabular().viewer().name("contact_mask_tabular"),
                 subscene_element_ids = info.subscene_element_ids().viewer().name("subscene_element_ids"),
                 subscene_mask_tabular = info.subscene_mask_tabular().viewer().name("subscene_mask_tabular"),
                 v2b = info.v2b().viewer().name("v2b"),
                 body_self_collision = info.body_self_collision().viewer().name("body_self_collision"),
                 d_hats = info.d_hats().viewer().name("d_hats"),
                 alpha  = alpha] __device__(IndexT i, IndexT j)
                {
                    const auto& codimV = codimVs(i);
                    const auto& E      = Es(j);

                    Vector3i cids = {contact_element_ids(codimV),
                                     contact_element_ids(E[0]),
                                     contact_element_ids(E[1])};

                    Vector3i scids = {subscene_element_ids(codimV),
                                      subscene_element_ids(E[0]),
                                      subscene_element_ids(E[1])};

                    // discard if the contact is disabled
                    if(!allow_PE_contact(subscene_mask_tabular, scids))
                        return false;
                    if(!allow_PE_contact(contact_mask_tabular, cids))
                        return false;

                    // discard if the vertex is on the edge
                    if(E[0] == codimV || E[1] == codimV)
                        return false;

                    auto body_i = v2b(codimV);
                    auto body_j = v2b(E[0]);
                    // skip self-collision for the same body if self collision off
                    if(body_i == body_j && !body_self_collision(body_i))
                        return false;

                    Vector3 E0  = Ps(E[0]);
                    Vector3 E1  = Ps(E[1]);
                    Vector3 dE0 = alpha * dxs(E[0]);
                    Vector3 dE1 = alpha * dxs(E[1]);

                    Vector3 P  = Ps(codimV);
                    Vector3 dP = alpha * dxs(codimV);

                    Float thickness = PE_thickness(thicknesses(codimV),
                                                   thicknesses(E[0]),
                                                   thicknesses(E[1]));
                    Float d_hat = PE_d_hat(d_hats(codimV), d_hats(E[0]), d_hats(E[1]));

                    Float expand = d_hat + thickness;

                    if(!distance::point_edge_ccd_broadphase(P, E0, E1, dP, dE0, dE1, expand))
                        return false;

                    return true;
                },
                candidate_CodimP_AllE_pairs);
        }
    }

    // Use AllE to query AllE
    {
        muda::KernelLabel label{__FUNCTION__, __FILE__, __LINE__};
        lbvh_E.detect(
            [Es          = Es.viewer().name("Es"),
             Ps          = Ps.viewer().name("Ps"),
             dxs         = dxs.viewer().name("dxs"),
             thicknesses = info.thicknesses().viewer().name("thicknesses"),
             contact_element_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
             contact_mask_tabular = info.contact_mask_tabular().viewer().name("contact_mask_tabular"),
             subscene_element_ids = info.subscene_element_ids().viewer().name("subscene_element_ids"),
             subscene_mask_tabular = info.subscene_mask_tabular().viewer().name("subscene_mask_tabular"),
             v2b = info.v2b().viewer().name("v2b"),
             body_self_collision = info.body_self_collision().viewer().name("body_self_collision"),
             d_hats = info.d_hats().viewer().name("d_hats"),
             alpha  = alpha] __device__(IndexT i, IndexT j)
            {
                const auto& E0 = Es(i);
                const auto& E1 = Es(j);

                Vector4i cids = {contact_element_ids(E0[0]),
                                 contact_element_ids(E0[1]),
                                 contact_element_ids(E1[0]),
                                 contact_element_ids(E1[1])};

                Vector4i scids = {subscene_element_ids(E0[0]),
                                  subscene_element_ids(E0[1]),
                                  subscene_element_ids(E1[0]),
                                  subscene_element_ids(E1[1])};

                // discard if the contact is disabled
                if(!allow_EE_contact(subscene_mask_tabular, scids))
                    return false;
                if(!allow_EE_contact(contact_mask_tabular, cids))
                    return false;

                // discard if the edges share same vertex
                if(E0[0] == E1[0] || E0[0] == E1[1] || E0[1] == E1[0] || E0[1] == E1[1])
                    return false;

                auto body_i = v2b(E0[0]);
                auto body_j = v2b(E1[0]);
                if(body_i == body_j && !body_self_collision(body_i))
                    return false;  // skip self-collision for the same body


                Vector3 E0_0  = Ps(E0[0]);
                Vector3 E0_1  = Ps(E0[1]);
                Vector3 dE0_0 = alpha * dxs(E0[0]);
                Vector3 dE0_1 = alpha * dxs(E0[1]);

                Vector3 E1_0  = Ps(E1[0]);
                Vector3 E1_1  = Ps(E1[1]);
                Vector3 dE1_0 = alpha * dxs(E1[0]);
                Vector3 dE1_1 = alpha * dxs(E1[1]);

                Float thickness = EE_thickness(thicknesses(E0[0]),
                                               thicknesses(E0[1]),
                                               thicknesses(E1[0]),
                                               thicknesses(E1[1]));

                Float d_hat =
                    EE_d_hat(d_hats(E0[0]), d_hats(E0[1]), d_hats(E1[0]), d_hats(E1[1]));

                Float expand = d_hat + thickness;

                if(!distance::edge_edge_ccd_broadphase(
                       E0_0, E0_1, E1_0, E1_1, dE0_0, dE0_1, dE1_0, dE1_1, expand))
                    return false;

                return true;
            },
            candidate_AllE_AllE_pairs);
    }

    // Use AllP to query AllT
    {
        muda::KernelLabel label{__FUNCTION__, __FILE__, __LINE__};
        lbvh_T.query(
            point_aabbs,
            [Vs          = Vs.viewer().name("Vs"),
             Fs          = Fs.viewer().name("Fs"),
             Ps          = Ps.viewer().name("Ps"),
             dxs         = dxs.viewer().name("dxs"),
             thicknesses = info.thicknesses().viewer().name("thicknesses"),
             contact_element_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
             contact_mask_tabular = info.contact_mask_tabular().viewer().name("contact_mask_tabular"),
             subscene_element_ids = info.subscene_element_ids().viewer().name("subscene_element_ids"),
             subscene_mask_tabular = info.subscene_mask_tabular().viewer().name("subscene_mask_tabular"),
             v2b = info.v2b().viewer().name("v2b"),
             body_self_collision = info.body_self_collision().viewer().name("body_self_collision"),
             d_hats = info.d_hats().viewer().name("d_hats"),
             alpha  = alpha] __device__(IndexT i, IndexT j)
            {
                auto V = Vs(i);
                auto F = Fs(j);

                Vector4i cids = {contact_element_ids(V),
                                 contact_element_ids(F[0]),
                                 contact_element_ids(F[1]),
                                 contact_element_ids(F[2])};

                Vector4i scids = {subscene_element_ids(V),
                                  subscene_element_ids(F[0]),
                                  subscene_element_ids(F[1]),
                                  subscene_element_ids(F[2])};

                // discard if the contact is disabled
                if(!allow_PT_contact(subscene_mask_tabular, scids))
                    return false;
                if(!allow_PT_contact(contact_mask_tabular, cids))
                    return false;

                // discard if the point is on the triangle
                if(F[0] == V || F[1] == V || F[2] == V)
                    return false;

                auto body_i = v2b(V);
                auto body_j = v2b(F[0]);
                // skip self-collision for the same body if self collision off
                if(body_i == body_j && !body_self_collision(body_i))
                    return false;


                Vector3 P  = Ps(V);
                Vector3 dP = alpha * dxs(V);

                Vector3 F0 = Ps(F[0]);
                Vector3 F1 = Ps(F[1]);
                Vector3 F2 = Ps(F[2]);

                Vector3 dF0 = alpha * dxs(F[0]);
                Vector3 dF1 = alpha * dxs(F[1]);
                Vector3 dF2 = alpha * dxs(F[2]);

                Float thickness = PT_thickness(thicknesses(V),
                                               thicknesses(F[0]),
                                               thicknesses(F[1]),
                                               thicknesses(F[2]));

                Float d_hat =
                    PT_d_hat(d_hats(V), d_hats(F[0]), d_hats(F[1]), d_hats(F[2]));

                Float expand = d_hat + thickness;

                if(!distance::point_triangle_ccd_broadphase(P, F0, F1, F2, dP, dF0, dF1, dF2, expand))
                    return false;

                return true;
            },
            candidate_AllP_AllT_pairs);
    }
}

void LBVHSimplexTrajectoryFilter::Impl::filter_active(FilterActiveInfo& info)
{
    using namespace muda;

    // we will filter-out the active pairs
    auto positions = info.positions();

    SizeT N_PCoimP  = candidate_AllP_CodimP_pairs.size();
    SizeT N_CodimPE = candidate_CodimP_AllE_pairs.size();
    SizeT N_PTs     = candidate_AllP_AllT_pairs.size();
    SizeT N_EEs     = candidate_AllE_AllE_pairs.size();

    // PT, EE, PT, PP can degenerate to PP
    temp_PPs.resize(N_PCoimP + N_CodimPE + N_PTs + N_EEs);
    // PT, EE, PT can degenerate to PE
    temp_PEs.resize(N_CodimPE + N_PTs + N_EEs);

    temp_PTs.resize(N_PTs);
    temp_EEs.resize(N_EEs);

    SizeT temp_PP_offset = 0;
    SizeT temp_PE_offset = 0;

    // AllP and CodimP
    if(N_PCoimP > 0)
    {
        auto PP_view = temp_PPs.view(temp_PP_offset, N_PCoimP);

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_AllP_CodimP_pairs.size(),
                   [positions = positions.viewer().name("positions"),
                    PCodimP_pairs = candidate_AllP_CodimP_pairs.viewer().name("PP_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    codim_vertices = info.codim_vertices().viewer().name("codim_vertices"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    temp_PPs = PP_view.viewer().name("temp_PPs"),
                    d_hats = info.d_hats().viewer().name("d_hats")] __device__(int i) mutable
                   {
                       // default invalid
                       auto& PP = temp_PPs(i);
                       PP.setConstant(-1);

                       Vector2i indices = PCodimP_pairs(i);

                       IndexT P0 = surf_vertices(indices(0));
                       IndexT P1 = codim_vertices(indices(1));


                       const auto& V0 = positions(P0);
                       const auto& V1 = positions(P1);

                       Float thickness = PP_thickness(thicknesses(P0), thicknesses(P1));
                       Float d_hat = PP_d_hat(d_hats(P0), d_hats(P1));

                       Vector2 range = D_range(thickness, d_hat);

                       Float D;
                       distance::point_point_distance2(V0, V1, D);


                       if(!is_active_D(range, D))
                           return;  // early return

                       PP = {P0, P1};
                   });

        temp_PP_offset += N_PCoimP;
    }
    // CodimP and AllE
    if(N_CodimPE > 0)
    {
        auto PP_view = temp_PPs.view(temp_PP_offset, N_CodimPE);
        auto PE_view = temp_PEs.view(temp_PE_offset, N_CodimPE);

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(
                candidate_CodimP_AllE_pairs.size(),
                [positions = positions.viewer().name("positions"),
                 CodimP_AllE_pairs = candidate_CodimP_AllE_pairs.viewer().name("PE_pairs"),
                 codim_veritces = info.codim_vertices().viewer().name("codim_vertices"),
                 surf_edges  = info.surf_edges().viewer().name("surf_edges"),
                 thicknesses = info.thicknesses().viewer().name("thicknesses"),
                 temp_PPs    = PP_view.viewer().name("temp_PPs"),
                 temp_PEs    = PE_view.viewer().name("temp_PEs"),
                 d_hats = info.d_hats().viewer().name("d_hats")] __device__(int i) mutable
                {
                    auto& PP = temp_PPs(i);
                    PP.setConstant(-1);
                    auto& PE = temp_PEs(i);
                    PE.setConstant(-1);

                    Vector2i indices = CodimP_AllE_pairs(i);
                    IndexT   V       = codim_veritces(indices(0));
                    Vector2i E       = surf_edges(indices(1));

                    Vector3i vIs = {V, E(0), E(1)};
                    Vector3 Ps[] = {positions(vIs(0)), positions(vIs(1)), positions(vIs(2))};

                    Float thickness = PE_thickness(
                        thicknesses(V), thicknesses(E(0)), thicknesses(E(1)));

                    Float d_hat = PE_d_hat(d_hats(V), d_hats(E(0)), d_hats(E(1)));


                    Vector3i flag =
                        distance::point_edge_distance_flag(Ps[0], Ps[1], Ps[2]);

                    Vector2 range = D_range(thickness, d_hat);

                    Float D;
                    distance::point_edge_distance2(flag, Ps[0], Ps[1], Ps[2], D);

                    if(!is_active_D(range, D))
                        return;  // early return

                    Vector3i offsets;
                    auto dim = distance::degenerate_point_edge(flag, offsets);

                    switch(dim)
                    {
                        case 2:  // PP
                        {
                            IndexT V0 = vIs(offsets(0));
                            IndexT V1 = vIs(offsets(1));
                            PP        = {V0, V1};
                        }
                        break;
                        case 3:  // PE
                        {
                            PE = vIs;
                        }
                        break;
                        default: {
                            MUDA_ERROR_WITH_LOCATION("unexpected degenerate case dim=%d", dim);
                        }
                        break;
                    }
                });

        temp_PP_offset += N_CodimPE;
        temp_PE_offset += N_CodimPE;
    }

    // AllP and AllT
    {
        auto PP_view = temp_PPs.view(temp_PP_offset, N_PTs);
        auto PE_view = temp_PEs.view(temp_PE_offset, N_PTs);

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(
                candidate_AllP_AllT_pairs.size(),
                [positions = positions.viewer().name("Ps"),
                 PT_pairs = candidate_AllP_AllT_pairs.viewer().name("PT_pairs"),
                 surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                 surf_triangles = info.surf_triangles().viewer().name("surf_triangles"),
                 thicknesses = info.thicknesses().viewer().name("thicknesses"),
                 temp_PPs    = PP_view.viewer().name("temp_PPs"),
                 temp_PEs    = PE_view.viewer().name("temp_PEs"),
                 temp_PTs    = temp_PTs.viewer().name("temp_PTs"),
                 d_hats = info.d_hats().viewer().name("d_hats")] __device__(int i) mutable
                {
                    auto& PP = temp_PPs(i);
                    PP.setConstant(-1);
                    auto& PE = temp_PEs(i);
                    PE.setConstant(-1);
                    auto& PT = temp_PTs(i);
                    PT.setConstant(-1);

                    Vector2i indices = PT_pairs(i);
                    IndexT   V       = surf_vertices(indices(0));
                    Vector3i F       = surf_triangles(indices(1));

                    Vector4i vIs  = {V, F(0), F(1), F(2)};
                    Vector3  Ps[] = {positions(vIs(0)),
                                     positions(vIs(1)),
                                     positions(vIs(2)),
                                     positions(vIs(3))};

                    Float thickness = PT_thickness(thicknesses(V),
                                                   thicknesses(F(0)),
                                                   thicknesses(F(1)),
                                                   thicknesses(F(2)));

                    Float d_hat =
                        PT_d_hat(d_hats(V), d_hats(F(0)), d_hats(F(1)), d_hats(F(2)));

                    Vector4i flag =
                        distance::point_triangle_distance_flag(Ps[0], Ps[1], Ps[2], Ps[3]);

                    Vector2 range = D_range(thickness, d_hat);

                    Float D;
                    distance::point_triangle_distance2(flag, Ps[0], Ps[1], Ps[2], Ps[3], D);

                    MUDA_ASSERT(
                        D > 0.0, "D=%f, V F = (%d,%d,%d,%d)", D, vIs(0), vIs(1), vIs(2), vIs(3));

                    if(!is_active_D(range, D))
                        return;  // early return

                    Vector4i offsets;
                    auto dim = distance::degenerate_point_triangle(flag, offsets);

                    switch(dim)
                    {
                        case 2:  // PP
                        {
                            IndexT V0 = vIs(offsets(0));
                            IndexT V1 = vIs(offsets(1));
                            PP        = {V0, V1};
                        }
                        break;
                        case 3:  // PE
                        {
                            IndexT V0 = vIs(offsets(0));
                            IndexT V1 = vIs(offsets(1));
                            IndexT V2 = vIs(offsets(2));
                            PE        = {V0, V1, V2};
                        }
                        break;
                        case 4:  // PT
                        {
                            PT = vIs;
                        }
                        break;
                        default: {
                            MUDA_ERROR_WITH_LOCATION("unexpected degenerate case dim=%d", dim);
                        }
                        break;
                    }
                });

        temp_PP_offset += N_PTs;
        temp_PE_offset += N_PTs;
    }
    // AllE and AllE
    {
        auto PP_view = temp_PPs.view(temp_PP_offset, N_EEs);
        auto PE_view = temp_PEs.view(temp_PE_offset, N_EEs);

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(
                candidate_AllE_AllE_pairs.size(),
                [positions = positions.viewer().name("Ps"),
                 rest_positions = info.rest_positions().viewer().name("rest_positions"),
                 EE_pairs = candidate_AllE_AllE_pairs.viewer().name("EE_pairs"),
                 surf_edges  = info.surf_edges().viewer().name("surf_edges"),
                 thicknesses = info.thicknesses().viewer().name("thicknesses"),
                 temp_PPs    = PP_view.viewer().name("temp_PPs"),
                 temp_PEs    = PE_view.viewer().name("temp_PEs"),
                 temp_EEs    = temp_EEs.viewer().name("temp_EEs"),
                 d_hats = info.d_hats().viewer().name("d_hats")] __device__(int i) mutable
                {
                    auto& PP = temp_PPs(i);
                    PP.setConstant(-1);
                    auto& PE = temp_PEs(i);
                    PE.setConstant(-1);
                    auto& EE = temp_EEs(i);
                    EE.setConstant(-1);

                    Vector2i indices = EE_pairs(i);
                    Vector2i E0      = surf_edges(indices(0));
                    Vector2i E1      = surf_edges(indices(1));

                    Vector4i vIs  = {E0(0), E0(1), E1(0), E1(1)};
                    Vector3  Ps[] = {positions(vIs(0)),
                                     positions(vIs(1)),
                                     positions(vIs(2)),
                                     positions(vIs(3))};

                    Float thickness = EE_thickness(thicknesses(E0(0)),
                                                   thicknesses(E0(1)),
                                                   thicknesses(E1(0)),
                                                   thicknesses(E1(1)));

                    Float d_hat = EE_d_hat(
                        d_hats(E0(0)), d_hats(E0(1)), d_hats(E1(0)), d_hats(E1(1)));

                    Vector2 range = D_range(thickness, d_hat);

                    Vector4i flag =
                        distance::edge_edge_distance_flag(Ps[0], Ps[1], Ps[2], Ps[3]);

                    Float D;
                    distance::edge_edge_distance2(flag, Ps[0], Ps[1], Ps[2], Ps[3], D);

                    if(!is_active_D(range, D))
                        return;  // early return

                    Float eps_x;
                    distance::edge_edge_mollifier_threshold(rest_positions(vIs(0)),
                                                            rest_positions(vIs(1)),
                                                            rest_positions(vIs(2)),
                                                            rest_positions(vIs(3)),
                                                            eps_x);

                    if(distance::need_mollify(Ps[0], Ps[1], Ps[2], Ps[3], eps_x))
                    {
                        EE = vIs;
                        return;
                    }
                    else  // classify to EE/PE/PP
                    {
                        Vector4i offsets;
                        auto dim = distance::degenerate_edge_edge(flag, offsets);

                        switch(dim)
                        {
                            case 2:  // PP
                            {
                                IndexT V0 = vIs(offsets(0));
                                IndexT V1 = vIs(offsets(1));
                                PP        = {V0, V1};
                            }
                            break;
                            case 3:  // PE
                            {
                                IndexT V0 = vIs(offsets(0));
                                IndexT V1 = vIs(offsets(1));
                                IndexT V2 = vIs(offsets(2));
                                PE        = {V0, V1, V2};
                            }
                            break;
                            case 4:  // EE
                            {
                                EE = vIs;
                            }
                            break;
                            default: {
                                MUDA_ERROR_WITH_LOCATION("unexpected degenerate case dim=%d", dim);
                            }
                            break;
                        }
                    }
                })
            .wait();

        temp_PP_offset += N_EEs;
        temp_PE_offset += N_EEs;
    }

    UIPC_ASSERT(temp_PP_offset == temp_PPs.size(), "size mismatch");
    UIPC_ASSERT(temp_PE_offset == temp_PEs.size(), "size mismatch");

    {  // select the valid ones
        PPs.resize(temp_PPs.size());
        PEs.resize(temp_PEs.size());
        PTs.resize(temp_PTs.size());
        EEs.resize(temp_EEs.size());

        DeviceSelect().If(temp_PPs.data(),
                          PPs.data(),
                          selected_PP_count.data(),
                          temp_PPs.size(),
                          [] HIPCUB_RUNTIME_FUNCTION(const Vector2i& PP)
                          { return PP(0) != -1; });

        DeviceSelect().If(temp_PEs.data(),
                          PEs.data(),
                          selected_PE_count.data(),
                          temp_PEs.size(),
                          [] HIPCUB_RUNTIME_FUNCTION(const Vector3i& PE)
                          { return PE(0) != -1; });

        DeviceSelect().If(temp_PTs.data(),
                          PTs.data(),
                          selected_PT_count.data(),
                          temp_PTs.size(),
                          [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& PT)
                          { return PT(0) != -1; });

        DeviceSelect().If(temp_EEs.data(),
                          EEs.data(),
                          selected_EE_count.data(),
                          temp_EEs.size(),
                          [] HIPCUB_RUNTIME_FUNCTION(const Vector4i& EE)
                          { return EE(0) != -1; });

        IndexT PP_count = selected_PP_count;
        IndexT PE_count = selected_PE_count;
        IndexT PT_count = selected_PT_count;
        IndexT EE_count = selected_EE_count;

        PPs.resize(PP_count);
        PEs.resize(PE_count);
        PTs.resize(PT_count);
        EEs.resize(EE_count);
    }

    info.PPs(PPs);
    info.PEs(PEs);
    info.PTs(PTs);
    info.EEs(EEs);

    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector2i> PPs_host;
        std::vector<Float>    PP_thicknesses_host;

        std::vector<Vector3i> PEs_host;
        std::vector<Float>    PE_thicknesses_host;

        std::vector<Vector4i> PTs_host;
        std::vector<Float>    PT_thicknesses_host;

        std::vector<Vector4i> EEs_host;
        std::vector<Float>    EE_thicknesses_host;

        PPs.copy_to(PPs_host);
        PEs.copy_to(PEs_host);
        PTs.copy_to(PTs_host);
        EEs.copy_to(EEs_host);

        std::cout << "filter result:" << std::endl;

        for(auto&& [PP, thickness] : zip(PPs_host, PP_thicknesses_host))
        {
            std::cout << "PP: " << PP.transpose() << " thickness: " << thickness << "\n";
        }

        for(auto&& [PE, thickness] : zip(PEs_host, PE_thicknesses_host))
        {
            std::cout << "PE: " << PE.transpose() << " thickness: " << thickness << "\n";
        }

        for(auto&& [PT, thickness] : zip(PTs_host, PT_thicknesses_host))
        {
            std::cout << "PT: " << PT.transpose() << " thickness: " << thickness << "\n";
        }

        for(auto&& [EE, thickness] : zip(EEs_host, EE_thicknesses_host))
        {
            std::cout << "EE: " << EE.transpose() << " thickness: " << thickness << "\n";
        }

        std::cout << std::flush;
    }
}

void LBVHSimplexTrajectoryFilter::Impl::filter_toi(FilterTOIInfo& info)
{
    using namespace muda;

    auto toi_size =
        candidate_AllP_CodimP_pairs.size() + candidate_CodimP_AllE_pairs.size()
        + candidate_AllP_AllT_pairs.size() + candidate_AllE_AllE_pairs.size();

    tois.resize(toi_size);

    auto offset  = 0;
    auto PP_tois = tois.view(offset, candidate_AllP_CodimP_pairs.size());
    offset += candidate_AllP_CodimP_pairs.size();
    auto PE_tois = tois.view(offset, candidate_CodimP_AllE_pairs.size());
    offset += candidate_CodimP_AllE_pairs.size();
    auto PT_tois = tois.view(offset, candidate_AllP_AllT_pairs.size());
    offset += candidate_AllP_AllT_pairs.size();
    auto EE_tois = tois.view(offset, candidate_AllE_AllE_pairs.size());
    offset += candidate_AllE_AllE_pairs.size();

    UIPC_ASSERT(offset == toi_size, "size mismatch");


    // TODO: Now hard code the minimum separation coefficient
    // gap = eta * (dist2_cur - thickness * thickness) / (dist_cur + thickness);
    constexpr Float eta = 0.1;

    // TODO: Now hard code the maximum iteration
    constexpr SizeT max_iter = 1000;

    // large enough toi (>1)
    constexpr Float large_enough_toi = 1.1;

    // AllP and CodimP
    {
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_AllP_CodimP_pairs.size(),
                   [PP_tois = PP_tois.viewer().name("PP_tois"),
                    PCodimP_pairs = candidate_AllP_CodimP_pairs.viewer().name("PP_pairs"),
                    codim_vertices = info.codim_vertices().viewer().name("codim_vertices"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    positions = info.positions().viewer().name("Ps"),
                    dxs       = info.displacements().viewer().name("dxs"),
                    d_hats    = info.d_hats().viewer().name("d_hats"),
                    alpha     = info.alpha(),

                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto   indices = PCodimP_pairs(i);
                       IndexT V0      = surf_vertices(indices(0));
                       IndexT V1      = codim_vertices(indices(1));

                       Float thickness = PP_thickness(thicknesses(V0), thicknesses(V1));
                       Float d_hat = PP_d_hat(d_hats(V0), d_hats(V1));

                       Vector3 VP0  = positions(V0);
                       Vector3 VP1  = positions(V1);
                       Vector3 dVP0 = alpha * dxs(V0);
                       Vector3 dVP1 = alpha * dxs(V1);

                       Float toi = large_enough_toi;

                       bool faraway = !distance::point_point_ccd_broadphase(
                           VP0, VP1, dVP0, dVP1, d_hat + thickness);

                       if(faraway)
                       {
                           PP_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::point_point_ccd(
                           VP0, VP1, dVP0, dVP1, eta, thickness, max_iter, toi);

                       if(!hit)
                           toi = large_enough_toi;

                       PP_tois(i) = toi;
                   });
    }

    // CodimP and AllE
    {
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_CodimP_AllE_pairs.size(),
                   [PE_tois = PE_tois.viewer().name("PE_tois"),
                    CodimP_AllE_pairs = candidate_CodimP_AllE_pairs.viewer().name("PE_pairs"),
                    codim_vertices = info.codim_vertices().viewer().name("codim_vertices"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    Ps         = info.positions().viewer().name("Ps"),
                    dxs        = info.displacements().viewer().name("dxs"),
                    d_hats     = info.d_hats().viewer().name("d_hats"),
                    alpha      = info.alpha(),
                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto     indices = CodimP_AllE_pairs(i);
                       IndexT   V       = codim_vertices(indices(0));
                       Vector2i E       = surf_edges(indices(1));

                       Float thickness = PE_thickness(
                           thicknesses(V), thicknesses(E(0)), thicknesses(E(1)));
                       Float d_hat = PE_d_hat(d_hats(V), d_hats(E(0)), d_hats(E(1)));

                       Vector3 VP  = Ps(V);
                       Vector3 dVP = alpha * dxs(V);

                       Vector3 EP0  = Ps(E[0]);
                       Vector3 EP1  = Ps(E[1]);
                       Vector3 dEP0 = alpha * dxs(E[0]);
                       Vector3 dEP1 = alpha * dxs(E[1]);

                       Float toi = large_enough_toi;

                       bool faraway = !distance::point_edge_ccd_broadphase(
                           VP, EP0, EP1, dVP, dEP0, dEP1, d_hat + thickness);

                       if(faraway)
                       {
                           PE_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::point_edge_ccd(
                           VP, EP0, EP1, dVP, dEP0, dEP1, eta, thickness, max_iter, toi);

                       if(!hit)
                           toi = large_enough_toi;

                       PE_tois(i) = toi;
                   });
    }

    // AllP and AllT
    {
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_AllP_AllT_pairs.size(),
                   [PT_tois = PT_tois.viewer().name("PT_tois"),
                    PT_pairs = candidate_AllP_AllT_pairs.viewer().name("PT_pairs"),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    surf_triangles = info.surf_triangles().viewer().name("surf_triangles"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    Ps     = info.positions().viewer().name("Ps"),
                    dxs    = info.displacements().viewer().name("dxs"),
                    d_hats = info.d_hats().viewer().name("d_hats"),
                    alpha  = info.alpha(),
                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto     indices = PT_pairs(i);
                       IndexT   V       = surf_vertices(indices(0));
                       Vector3i F       = surf_triangles(indices(1));

                       Float thickness = PT_thickness(thicknesses(V),
                                                      thicknesses(F(0)),
                                                      thicknesses(F(1)),
                                                      thicknesses(F(2)));
                       Float d_hat =
                           PT_d_hat(d_hats(V), d_hats(F(0)), d_hats(F(1)), d_hats(F(2)));

                       Vector3 VP  = Ps(V);
                       Vector3 dVP = alpha * dxs(V);

                       Vector3 FP0 = Ps(F[0]);
                       Vector3 FP1 = Ps(F[1]);
                       Vector3 FP2 = Ps(F[2]);

                       Vector3 dFP0 = alpha * dxs(F[0]);
                       Vector3 dFP1 = alpha * dxs(F[1]);
                       Vector3 dFP2 = alpha * dxs(F[2]);

                       Float toi = large_enough_toi;


                       bool faraway = !distance::point_triangle_ccd_broadphase(
                           VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, d_hat + thickness);

                       if(faraway)
                       {
                           PT_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::point_triangle_ccd(
                           VP, FP0, FP1, FP2, dVP, dFP0, dFP1, dFP2, eta, thickness, max_iter, toi);

                       if(!hit)
                           toi = large_enough_toi;

                       PT_tois(i) = toi;
                   });
    }

    // AllE and AllE
    {
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(candidate_AllE_AllE_pairs.size(),
                   [EE_tois = EE_tois.viewer().name("EE_tois"),
                    EE_pairs = candidate_AllE_AllE_pairs.viewer().name("EE_pairs"),
                    surf_edges = info.surf_edges().viewer().name("surf_edges"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    Ps     = info.positions().viewer().name("Ps"),
                    dxs    = info.displacements().viewer().name("dxs"),
                    d_hats = info.d_hats().viewer().name("d_hats"),
                    alpha  = info.alpha(),
                    eta,
                    max_iter,
                    large_enough_toi] __device__(int i) mutable
                   {
                       auto     indices = EE_pairs(i);
                       Vector2i E0      = surf_edges(indices(0));
                       Vector2i E1      = surf_edges(indices(1));

                       Float thickness = EE_thickness(thicknesses(E0(0)),
                                                      thicknesses(E0(1)),
                                                      thicknesses(E1(0)),
                                                      thicknesses(E1(1)));

                       Float d_hat = EE_d_hat(
                           d_hats(E0(0)), d_hats(E0(1)), d_hats(E1(0)), d_hats(E1(1)));


                       Vector3 EP0  = Ps(E0[0]);
                       Vector3 EP1  = Ps(E0[1]);
                       Vector3 dEP0 = alpha * dxs(E0[0]);
                       Vector3 dEP1 = alpha * dxs(E0[1]);

                       Vector3 EP2  = Ps(E1[0]);
                       Vector3 EP3  = Ps(E1[1]);
                       Vector3 dEP2 = alpha * dxs(E1[0]);
                       Vector3 dEP3 = alpha * dxs(E1[1]);

                       Float toi = large_enough_toi;

                       bool faraway = !distance::edge_edge_ccd_broadphase(
                           // position
                           EP0,
                           EP1,
                           EP2,
                           EP3,
                           // displacement
                           dEP0,
                           dEP1,
                           dEP2,
                           dEP3,
                           d_hat + thickness);

                       if(faraway)
                       {
                           EE_tois(i) = toi;
                           return;
                       }

                       bool hit = distance::edge_edge_ccd(
                           // position
                           EP0,
                           EP1,
                           EP2,
                           EP3,
                           // displacement
                           dEP0,
                           dEP1,
                           dEP2,
                           dEP3,
                           eta,
                           thickness,
                           max_iter,
                           toi);

                       if(!hit)
                           toi = large_enough_toi;

                       EE_tois(i) = toi;
                   });
    }

    if(tois.size())
    {
        // get min toi
        DeviceReduce().Min(tois.data(), info.toi().data(), tois.size());
    }
    else
    {
        info.toi().fill(large_enough_toi);
    }
}
}  // namespace uipc::backend::cuda
