#include "hip/hip_runtime.h"
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_trajectory_filter.h>
#include <sim_engine.h>
#include <contact_system/contact_reporter.h>
#include <uipc/common/enumerate.h>
#include <kernel_cout.h>
#include <uipc/common/unit.h>
#include <uipc/common/zip.h>

namespace uipc::backend
{
template <>
class SimSystemCreator<cuda::GlobalContactManager>
{
  public:
    static U<cuda::GlobalContactManager> create(cuda::SimEngine& engine)
    {
        auto contact_enable_attr =
            engine.world().scene().config().find<IndexT>("contact/enable");
        bool contact_enable = contact_enable_attr->view()[0] != 0;

        auto& types = engine.world().scene().constitution_tabular().types();
        bool  has_inter_primitive_constitution =
            types.find(std::string{builtin::InterPrimitive}) != types.end();

        if(contact_enable || has_inter_primitive_constitution)
            return make_unique<cuda::GlobalContactManager>(engine);
        return nullptr;
    }
};
}  // namespace uipc::backend

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalContactManager);

void GlobalContactManager::do_build()
{
    const auto& config = world().scene().config();

    m_impl.global_vertex_manager    = require<GlobalVertexManager>();
    m_impl.global_trajectory_filter = find<GlobalTrajectoryFilter>();


    auto d_hat_attr = config.find<Float>("contact/d_hat");
    m_impl.d_hat    = d_hat_attr->view()[0];

    auto dt_attr = config.find<Float>("dt");
    m_impl.dt    = dt_attr->view()[0];

    auto eps_velocity_attr = config.find<Float>("contact/eps_velocity");
    m_impl.eps_velocity    = eps_velocity_attr->view()[0];

    auto cfl_enable_attr = config.find<IndexT>("cfl/enable");
    m_impl.cfl_enabled   = cfl_enable_attr->view()[0] != 0;

    m_impl.kappa = world().scene().contact_tabular().default_model().resistance();
}

muda::CBuffer2DView<IndexT> GlobalContactManager::contact_mask_tabular() const noexcept
{
    return m_impl.contact_mask_tabular;
}

muda::CBuffer2DView<IndexT> GlobalContactManager::subscene_mask_tabular() const noexcept
{
    return m_impl.subscene_mask_tabular;
}

void GlobalContactManager::Impl::init(WorldVisitor& world)
{
    // 1) init tabular
    _build_contact_tabular(world);
    _build_subscene_tabular(world);


    // 2) vertex contact info
    vert_is_active_contact.resize(global_vertex_manager->positions().size(), 0);
    vert_disp_norms.resize(global_vertex_manager->positions().size(), 0.0);

    // 3) reporters
    auto contact_reporter_view = contact_reporters.view();
    for(auto&& [i, R] : enumerate(contact_reporter_view))
        R->init();
    for(auto&& [i, R] : enumerate(contact_reporter_view))
        R->m_index = i;
}

using MaskMatrix = Eigen::Matrix<IndexT, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>;

void GlobalContactManager::Impl::_build_contact_tabular(WorldVisitor& world)
{
    auto contact_models = world.scene().contact_tabular().contact_models();

    auto attr_topo          = contact_models.find<Vector2i>("topo");
    auto attr_resistance    = contact_models.find<Float>("resistance");
    auto attr_friction_rate = contact_models.find<Float>("friction_rate");
    auto attr_enabled       = contact_models.find<IndexT>("is_enabled");

    UIPC_ASSERT(attr_topo != nullptr, "topo is not found in contact tabular");
    UIPC_ASSERT(attr_resistance != nullptr, "resistance is not found in contact tabular");
    UIPC_ASSERT(attr_friction_rate != nullptr, "friction_rate is not found in contact tabular");
    UIPC_ASSERT(attr_enabled != nullptr, "is_enabled is not found in contact tabular");

    auto topo_view          = attr_topo->view();
    auto resistance_view    = attr_resistance->view();
    auto friction_rate_view = attr_friction_rate->view();
    auto enabled_view       = attr_enabled->view();

    auto N = world.scene().contact_tabular().element_count();

    // default turn on the contact between two same types
    h_contact_mask_tabular.resize(N * N, 1);

    auto mask_map = Eigen::Map<MaskMatrix>(h_contact_mask_tabular.data(), N, N);

    h_contact_tabular.resize(
        N * N, ContactCoeff{.kappa = resistance_view[0], .mu = friction_rate_view[0]});

    for(auto&& [ids, kappa, mu, is_enabled] :
        zip(topo_view, resistance_view, friction_rate_view, enabled_view))
    {
        ContactCoeff coeff{.kappa = kappa, .mu = mu};

        auto upper                 = ids.x() * N + ids.y();
        h_contact_tabular[upper]   = coeff;
        mask_map(ids.x(), ids.y()) = is_enabled;


        auto lower                 = ids.y() * N + ids.x();
        h_contact_tabular[lower]   = coeff;
        mask_map(ids.y(), ids.x()) = is_enabled;
    }

    contact_tabular.resize(muda::Extent2D{N, N});
    contact_tabular.view().copy_from(h_contact_tabular.data());

    contact_mask_tabular.resize(muda::Extent2D{N, N});
    contact_mask_tabular.view().copy_from(h_contact_mask_tabular.data());
}

void GlobalContactManager::Impl::_build_subscene_tabular(WorldVisitor& world)
{
    auto subscene_models = world.scene().subscene_tabular().subscene_models();

    auto topo       = subscene_models.find<Vector2i>("topo");
    auto is_enabled = subscene_models.find<IndexT>("is_enabled");


    UIPC_ASSERT(topo != nullptr, "subscene topo is not found in contact tabular");
    UIPC_ASSERT(is_enabled != nullptr, "subscene is_enabled is not found in contact tabular");

    auto topo_view   = topo->view();
    auto enable_view = is_enabled->view();
    auto SN          = world.scene().subscene_tabular().element_count();

    h_subcene_mask_tabular.resize(SN * SN);
    auto mask_map = Eigen::Map<MaskMatrix>(h_subcene_mask_tabular.data(), SN, SN);
    // default turn off the contact between two different subscenes
    mask_map.setIdentity();  // enable self-scene-contact

    for(auto&& [ids, is_enabled] : zip(topo_view, enable_view))
    {
        mask_map(ids.x(), ids.y()) = is_enabled;
        mask_map(ids.y(), ids.x()) = is_enabled;
    }

    subscene_mask_tabular.resize(muda::Extent2D{SN, SN});
    subscene_mask_tabular.view().copy_from(h_subcene_mask_tabular.data());
}

void GlobalContactManager::Impl::compute_d_hat()
{
    // TODO: Now do nothing
}

void GlobalContactManager::Impl::compute_adaptive_kappa()
{
    // TODO: Now do nothing
}

Float GlobalContactManager::Impl::compute_cfl_condition()
{
    if(!cfl_enabled)  // if cfl is disabled, just return 1.0
        return 1.0;

    vert_is_active_contact.fill(0);  // clear the active flag

    if(global_trajectory_filter)
    {
        global_trajectory_filter->label_active_vertices();

        auto displacements = global_vertex_manager->displacements();

        using namespace muda;
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(displacements.size(),
                   [disps      = displacements.cviewer().name("disp"),
                    disp_norms = vert_disp_norms.viewer().name("disp_norm"),
                    is_contact_active = vert_is_active_contact.viewer().name(
                        "vert_is_contact_active")] __device__(int i) mutable
                   {
                       // if the contact is not active, then the displacement is ignored
                       disp_norms(i) = is_contact_active(i) ? disps(i).norm() : 0.0;
                   });

        DeviceReduce().Max(vert_disp_norms.data(),
                           max_disp_norm.data(),
                           vert_disp_norms.size());

        Float h_max_disp_norm = max_disp_norm;
        return h_max_disp_norm == 0.0 ? 1.0 : std::min(0.5 * d_hat / h_max_disp_norm, 1.0);
    }
    else
    {
        return 1.0;
    }
}
}  // namespace uipc::backend::cuda


namespace uipc::backend::cuda
{
void GlobalContactManager::compute_d_hat()
{
    m_impl.compute_d_hat();
}

void GlobalContactManager::compute_adaptive_kappa()
{
    m_impl.compute_adaptive_kappa();
}

Float GlobalContactManager::compute_cfl_condition()
{
    return m_impl.compute_cfl_condition();
}

void GlobalContactManager::init()
{
    m_impl.init(world());
}

Float GlobalContactManager::d_hat() const
{
    return m_impl.d_hat;
}
Float GlobalContactManager::eps_velocity() const
{
    return m_impl.eps_velocity;
}
bool GlobalContactManager::cfl_enabled() const
{
    return m_impl.cfl_enabled;
}

void GlobalContactManager::add_reporter(ContactReporter* reporter)
{
    check_state(SimEngineState::BuildSystems, "add_reporter()");
    UIPC_ASSERT(reporter != nullptr, "reporter is nullptr");
    m_impl.contact_reporters.register_subsystem(*reporter);
}

muda::CBuffer2DView<ContactCoeff> GlobalContactManager::contact_tabular() const noexcept
{
    return m_impl.contact_tabular;
}
}  // namespace uipc::backend::cuda