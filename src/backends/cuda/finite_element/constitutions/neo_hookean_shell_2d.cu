#include <finite_element/codim_2d_constitution.h>
#include <finite_element/constitutions/neo_hookean_shell_2d_function.h>
#include <kernel_cout.h>
#include <muda/ext/eigen/log_proxy.h>
#include <Eigen/Dense>
#include <muda/ext/eigen/inverse.h>
#include <utils/codim_thickness.h>
#include <utils/make_spd.h>
#include <utils/matrix_assembler.h>

namespace uipc::backend::cuda
{
class NeoHookeanShell2D final : public Codim2DConstitution
{
  public:
    // Constitution UID by libuipc specification
    static constexpr U64 ConstitutionUID = 11;

    using Codim2DConstitution::Codim2DConstitution;

    vector<Float> h_kappas;
    vector<Float> h_lambdas;

    muda::DeviceBuffer<Float> kappas;
    muda::DeviceBuffer<Float> lambdas;

    virtual U64 get_uid() const noexcept override { return ConstitutionUID; }

    virtual void do_build(BuildInfo& info) override {}

    virtual void do_init(FiniteElementMethod::FilteredInfo& info) override
    {
        using ForEachInfo = FiniteElementMethod::ForEachInfo;

        auto geo_slots = world().scene().geometries();

        auto N = info.primitive_count();

        h_kappas.resize(N);
        h_lambdas.resize(N);

        info.for_each(
            geo_slots,
            [](geometry::SimplicialComplex& sc) -> auto
            {
                auto mu     = sc.triangles().find<Float>("mu");
                auto lambda = sc.triangles().find<Float>("lambda");

                return zip(mu->view(), lambda->view());
            },
            [&](const ForEachInfo& I, auto mu_and_lambda)
            {
                auto vI = I.global_index();

                auto&& [mu, lambda] = mu_and_lambda;
                h_kappas[vI]        = mu;
                h_lambdas[vI]       = lambda;
            });

        kappas.resize(N);
        kappas.view().copy_from(h_kappas.data());

        lambdas.resize(N);
        lambdas.view().copy_from(h_lambdas.data());
    }

    virtual void do_compute_energy(ComputeEnergyInfo& info) override
    {
        using namespace muda;
        namespace NH = sym::shell_neo_hookean_2d;

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(info.indices().size(),
                   [mus        = kappas.cviewer().name("mus"),
                    lambdas    = lambdas.cviewer().name("lambdas"),
                    rest_areas = info.rest_areas().viewer().name("rest_area"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    energies = info.energies().viewer().name("energies"),
                    indices  = info.indices().viewer().name("indices"),
                    xs       = info.xs().viewer().name("xs"),
                    x_bars   = info.x_bars().viewer().name("x_bars"),
                    dt       = info.dt()] __device__(int I)
                   {
                       Vector9  X;
                       Vector3i idx = indices(I);
                       for(int i = 0; i < 3; ++i)
                           X.segment<3>(3 * i) = xs(idx(i));

                       Vector9 X_bar;
                       for(int i = 0; i < 3; ++i)
                           X_bar.segment<3>(3 * i) = x_bars(idx(i));

                       Matrix2x2 IB;
                       NH::A(IB, X_bar);
                       IB = muda::eigen::inverse(IB);

                       if constexpr(RUNTIME_CHECK)
                       {
                           Matrix2x2 A;
                           NH::A(A, X);
                           Float detA = A.determinant();
                       }

                       Float mu        = mus(I);
                       Float lambda    = lambdas(I);
                       Float rest_area = rest_areas(I);
                       Float thickness = triangle_thickness(thicknesses(idx(0)),
                                                            thicknesses(idx(1)),
                                                            thicknesses(idx(2)));

                       Float E;
                       NH::E(E, mu, lambda, X, IB);
                       energies(I) = E * rest_area * thickness * dt * dt;
                   });
    }

    virtual void do_compute_gradient_hessian(ComputeGradientHessianInfo& info) override
    {
        using namespace muda;
        namespace NH = sym::shell_neo_hookean_2d;

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(info.indices().size(),
                   [mus     = kappas.cviewer().name("mus"),
                    lambdas = lambdas.cviewer().name("lambdas"),
                    indices = info.indices().viewer().name("indices"),
                    xs      = info.xs().viewer().name("xs"),
                    x_bars  = info.x_bars().viewer().name("x_bars"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    G3s        = info.gradients().viewer().name("gradients"),
                    H3x3s      = info.hessians().viewer().name("hessians"),
                    rest_areas = info.rest_areas().viewer().name("volumes"),
                    dt         = info.dt()] __device__(int I) mutable
                   {
                       Vector9  X;
                       Vector3i idx = indices(I);
                       for(int i = 0; i < 3; ++i)
                           X.segment<3>(3 * i) = xs(idx(i));

                       Vector9 X_bar;
                       for(int i = 0; i < 3; ++i)
                           X_bar.segment<3>(3 * i) = x_bars(idx(i));

                       Matrix2x2 IB;
                       NH::A(IB, X_bar);
                       IB = muda::eigen::inverse(IB);

                       Float mu        = mus(I);
                       Float lambda    = lambdas(I);
                       Float rest_area = rest_areas(I);
                       Float thickness = triangle_thickness(thicknesses(idx(0)),
                                                            thicknesses(idx(1)),
                                                            thicknesses(idx(2)));

                       Float Vdt2 = rest_area * thickness * dt * dt;

                       Vector9 G;
                       NH::dEdX(G, mu, lambda, X, IB);
                       G *= Vdt2;
                       DoubletVectorAssembler DVA{G3s};
                       DVA.segment<3>(I * 3).write(idx, G);

                       Matrix9x9 H;
                       NH::ddEddX(H, mu, lambda, X, IB);
                       H *= Vdt2;
                       make_spd(H);
                       TripletMatrixAssembler TMA{H3x3s};
                       TMA.block<3, 3>(I * 3 * 3).write(idx, H);
                   });
    }
};

REGISTER_SIM_SYSTEM(NeoHookeanShell2D);
}  // namespace uipc::backend::cuda
