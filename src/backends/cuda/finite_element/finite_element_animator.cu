#include <finite_element/finite_element_animator.h>
#include <finite_element/finite_element_constraint.h>
#include <uipc/builtin/attribute_name.h>
#include <uipc/common/enumerate.h>
#include <muda/cub/device/device_reduce.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(FiniteElementAnimator);

void FiniteElementAnimator::do_build(BuildInfo& info)
{
    m_impl.finite_element_method = &require<FiniteElementMethod>();
    m_impl.global_animator       = &require<GlobalAnimator>();
}

void FiniteElementAnimator::add_constraint(FiniteElementConstraint* constraint)
{
    m_impl.constraints.register_subsystem(*constraint);
}

void FiniteElementAnimator::assemble(AssembleInfo& info)
{
    // compute the gradient and hessian
    for(auto constraint : m_impl.constraints.view())
    {
        ComputeGradientHessianInfo this_info{
            &m_impl, constraint->m_index, info.dt(), info.hessians()};
        constraint->compute_gradient_hessian(this_info);
    }

    // assemble the gradient and hessian
    m_impl.assemble(info);
}

void FiniteElementAnimator::do_init()
{
    m_impl.init(world());
}

void FiniteElementAnimator::do_step()
{
    m_impl.step();
}

void FiniteElementAnimator::Impl::init(backend::WorldVisitor& world)
{
    // sort the constraints by uid
    auto constraint_view = constraints.view();

    std::sort(constraint_view.begin(),
              constraint_view.end(),
              [](const FiniteElementConstraint* a, const FiniteElementConstraint* b)
              { return a->uid() < b->uid(); });

    // setup constraint index and the mapping from uid to index
    for(auto&& [i, constraint] : enumerate(constraint_view))
    {
        auto uid                     = constraint->uid();
        uid_to_constraint_index[uid] = i;
        constraint->m_index          = i;
    }

    // +1 for total count
    constraint_geo_info_counts.resize(constraint_view.size() + 1, 0);
    constraint_geo_info_offsets.resize(constraint_view.size() + 1, 0);

    auto        geo_slots = world.scene().geometries();
    const auto& geo_infos = finite_element_method->m_impl.geo_infos;

    for(auto& info : geo_infos)
    {
        auto  geo_slot = geo_slots[info.geo_slot_index];
        auto& geo      = geo_slot->geometry();
        auto  uid      = geo.meta().find<U64>(builtin::constraint_uid);
        if(uid)
        {
            auto uid_value = uid->view().front();
            auto it        = uid_to_constraint_index.find(uid_value);
            UIPC_ASSERT(it != uid_to_constraint_index.end(),
                        "FiniteElementAnimator: Constraint uid not found");
            auto index = it->second;
            constraint_geo_info_counts[index]++;
        }
    }

    std::exclusive_scan(constraint_geo_info_counts.begin(),
                        constraint_geo_info_counts.end(),
                        constraint_geo_info_offsets.begin(),
                        0);

    auto total_anim_geo_info = constraint_geo_info_offsets.back();
    anim_geo_infos.resize(total_anim_geo_info);

    vector<SizeT> anim_geo_info_counter(constraint_view.size(), 0);

    for(auto& info : geo_infos)
    {
        auto  geo_slot = geo_slots[info.geo_slot_index];
        auto& geo      = geo_slot->geometry();
        auto  uid      = geo.meta().find<U64>(builtin::constraint_uid);
        if(uid)
        {
            auto uid_value = uid->view().front();
            auto it        = uid_to_constraint_index.find(uid_value);
            UIPC_ASSERT(it != uid_to_constraint_index.end(),
                        "Constraint: Constraint uid not found");
            auto index = it->second;
            auto offset =
                constraint_geo_info_offsets[index] + anim_geo_info_counter[index];
            anim_geo_infos[offset] = info;
            anim_geo_info_counter[index]++;
        }
    }

    vector<list<IndexT>> constraint_vertex_indices(constraint_view.size());
    for(auto& c : constraint_view)
    {
        auto constraint_geo_infos =
            span{anim_geo_infos}.subspan(constraint_geo_info_offsets[c->m_index],
                                         constraint_geo_info_counts[c->m_index]);

        auto& indices = constraint_vertex_indices[c->m_index];

        for(auto& info : constraint_geo_infos)
        {
            for(int i = 0; i < info.vertex_count; i++)
            {
                indices.push_back(info.vertex_offset + i);
            }
        }
    }

    constraint_vertex_counts.resize(constraint_view.size() + 1, 0);
    constraint_vertex_offsets.resize(constraint_view.size() + 1, 0);

    std::ranges::transform(constraint_vertex_indices,
                           constraint_vertex_counts.begin(),
                           [](const auto& indices) { return indices.size(); });

    std::exclusive_scan(constraint_vertex_counts.begin(),
                        constraint_vertex_counts.end(),
                        constraint_vertex_offsets.begin(),
                        0);

    auto total_vertex_count = constraint_vertex_offsets.back();
    anim_indices.resize(total_vertex_count);

    // expand the indices
    for(auto&& [i, indices] : enumerate(constraint_vertex_indices))
    {
        auto offset = constraint_vertex_offsets[i];
        for(auto& index : indices)
        {
            anim_indices[offset++] = index;
        }
    }

    // initialize the constraints
    for(auto constraint : constraint_view)
    {
        FilteredInfo info{this, constraint->m_index};
        constraint->init(info);
    }

    // reserve offsets and counts for constraints (+1 for total count)
    constraint_energy_offsets.resize(constraint_view.size() + 1, 0);
    constraint_energy_counts.resize(constraint_view.size() + 1, 0);
    constraint_gradient_offsets.resize(constraint_view.size() + 1, 0);
    constraint_gradient_counts.resize(constraint_view.size() + 1, 0);
    constraint_hessian_offsets.resize(constraint_view.size() + 1, 0);
    constraint_hessian_counts.resize(constraint_view.size() + 1, 0);
}

void FiniteElementAnimator::report_extent(ExtentInfo& info)
{
    info.hessian_block_count = m_impl.constraint_hessian_offsets.back();
}

void FiniteElementAnimator::Impl::step()
{
    for(auto constraint : constraints.view())
    {
        FilteredInfo info{this, constraint->m_index};
        constraint->step(info);
    }

    SizeT H3x3_count = 0;
    SizeT G3_count   = 0;
    SizeT E_count    = 0;

    // clear the last element
    constraint_energy_counts.back()   = 0;
    constraint_gradient_counts.back() = 0;
    constraint_hessian_counts.back()  = 0;

    for(auto&& [i, constraint] : enumerate(constraints.view()))
    {
        ReportExtentInfo this_info;
        constraint->report_extent(this_info);

        constraint_energy_counts[i]   = this_info.m_energy_count;
        constraint_gradient_counts[i] = this_info.m_gradient_segment_count;
        constraint_hessian_counts[i]  = this_info.m_hessian_block_count;
    }

    // update the offsets
    std::exclusive_scan(constraint_energy_counts.begin(),
                        constraint_energy_counts.end(),
                        constraint_energy_offsets.begin(),
                        0);

    E_count = constraint_energy_offsets.back();

    std::exclusive_scan(constraint_gradient_counts.begin(),
                        constraint_gradient_counts.end(),
                        constraint_gradient_offsets.begin(),
                        0);

    G3_count = constraint_gradient_offsets.back();

    std::exclusive_scan(constraint_hessian_counts.begin(),
                        constraint_hessian_counts.end(),
                        constraint_hessian_offsets.begin(),
                        0);

    H3x3_count = constraint_hessian_offsets.back();

    // resize the buffers
    IndexT vertex_count = finite_element_method->xs().size();
    constraint_energies.resize(E_count);
    constraint_gradient.resize(vertex_count, G3_count);
    constraint_hessian.resize(vertex_count, vertex_count, H3x3_count);
}

void FiniteElementAnimator::Impl::assemble(AssembleInfo& info)
{
    using namespace muda;

    // only need to setup gradient (from doublet vector to dense vector)
    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(constraint_gradient.doublet_count(),
               [anim_gradients = std::as_const(constraint_gradient).viewer().name("aim_gradients"),
                gradients = info.gradients().viewer().name("gradients"),
                is_fixed = fem().is_fixed.cviewer().name("is_fixed")] __device__(int I) mutable
               {
                   const auto& [i, G3] = anim_gradients(I);
                   if(is_fixed(i))
                   {
                       //
                   }
                   else
                   {
                       gradients.segment<3>(i * 3).atomic_add(G3);
                   }
               });
}

Float FiniteElementAnimator::compute_energy(LineSearcher::EnergyInfo& info)
{
    using namespace muda;
    for(auto constraint : m_impl.constraints.view())
    {
        ComputeEnergyInfo this_info{&m_impl, constraint->m_index, info.dt()};
        constraint->compute_energy(this_info);
    }

    DeviceReduce().Sum(m_impl.constraint_energies.data(),
                       m_impl.constraint_energy.data(),
                       m_impl.constraint_energies.size());

    // copy back to host
    Float E = m_impl.constraint_energy;

    return E;
}

auto FiniteElementAnimator::FilteredInfo::anim_geo_infos() const -> span<const AnimatedGeoInfo>
{
    return span<const AnimatedGeoInfo>{m_impl->anim_geo_infos}.subspan(
        m_impl->constraint_geo_info_offsets[m_index],
        m_impl->constraint_geo_info_counts[m_index]);
}

SizeT FiniteElementAnimator::FilteredInfo::anim_vertex_count() const noexcept
{
    return m_impl->constraint_vertex_counts[m_index];
}

span<const IndexT> FiniteElementAnimator::FilteredInfo::anim_indices() const
{
    auto offset = m_impl->constraint_vertex_offsets[m_index];
    auto count  = m_impl->constraint_vertex_counts[m_index];
    return span{m_impl->anim_indices}.subspan(offset, count);
}

Float FiniteElementAnimator::BaseInfo::substep_ratio() const noexcept
{
    return m_impl->global_animator->substep_ratio();
}

muda::CBufferView<Vector3> FiniteElementAnimator::BaseInfo::xs() const noexcept
{
    return m_impl->finite_element_method->xs();
}

muda::CBufferView<Vector3> FiniteElementAnimator::BaseInfo::x_prevs() const noexcept
{
    return m_impl->finite_element_method->x_prevs();
}

muda::CBufferView<Float> FiniteElementAnimator::BaseInfo::masses() const noexcept
{
    return m_impl->finite_element_method->masses();
}

muda::CBufferView<IndexT> FiniteElementAnimator::BaseInfo::is_fixed() const noexcept
{
    return m_impl->finite_element_method->is_fixed();
}

muda::BufferView<Float> FiniteElementAnimator::ComputeEnergyInfo::energies() const noexcept
{
    auto offset = m_impl->constraint_energy_offsets[m_index];
    auto count  = m_impl->constraint_energy_counts[m_index];
    return m_impl->constraint_energies.view(offset, count);
}

muda::DoubletVectorView<Float, 3> FiniteElementAnimator::ComputeGradientHessianInfo::gradients() const noexcept
{
    auto offset = m_impl->constraint_gradient_offsets[m_index];
    auto count  = m_impl->constraint_gradient_counts[m_index];
    return m_impl->constraint_gradient.view().subview(offset, count);
}

void FiniteElementAnimator::ReportExtentInfo::hessian_block_count(SizeT count) noexcept
{
    m_hessian_block_count = count;
}
void FiniteElementAnimator::ReportExtentInfo::gradient_segment_count(SizeT count) noexcept
{
    m_gradient_segment_count = count;
}
void FiniteElementAnimator::ReportExtentInfo::energy_count(SizeT count) noexcept
{
    m_energy_count = count;
}
}  // namespace uipc::backend::cuda
