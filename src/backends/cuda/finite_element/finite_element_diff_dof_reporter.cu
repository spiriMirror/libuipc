#include <finite_element/finite_element_diff_dof_reporter.h>

namespace uipc::backend::cuda
{
void FiniteElementDiffDofReporter::do_build(DiffDofReporter::BuildInfo& info)
{
    m_impl.fem   = &require<FiniteElementMethod>();
    auto dt_attr = world().scene().config().find<Float>("dt");
    m_impl.dt    = dt_attr->view()[0];
    BuildInfo this_info;
    do_build(this_info);
}

void FiniteElementDiffDofReporter::do_assemble(GlobalDiffSimManager::DiffDofInfo& info)
{
    DiffDofInfo this_info{&m_impl, info, m_impl.dt};
    do_assemble(this_info);
}

SizeT FiniteElementDiffDofReporter::DiffDofInfo::frame() const
{
    return m_global_info.frame();
}

IndexT FiniteElementDiffDofReporter::DiffDofInfo::dof_offset(SizeT frame) const
{
    // Frame Dof Offset + FEM Dof Offset => Frame FEM Dof Offset
    return m_global_info.dof_offset(frame) + m_impl->fem->dof_offset(frame);
}

IndexT FiniteElementDiffDofReporter::DiffDofInfo::dof_count(SizeT frame) const
{
    // FEM Dof Count => Frame FEM Dof Count
    return m_impl->fem->dof_count(frame);
}

muda::TripletMatrixView<Float, 1> FiniteElementDiffDofReporter::DiffDofInfo::H() const
{
    return m_global_info.H();
}

Float FiniteElementDiffDofReporter::DiffDofInfo::dt() const
{
    return m_dt;
}
}  // namespace uipc::backend::cuda