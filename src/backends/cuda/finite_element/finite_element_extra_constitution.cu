#include <finite_element/finite_element_extra_constitution.h>
#include <uipc/builtin/attribute_name.h>

namespace uipc::backend::cuda
{
void FiniteElementExtraConstitution::do_build(FiniteElementEnergyProducer::BuildInfo& info)
{
    m_impl.finite_element_method = &require<FiniteElementMethod>();

    auto uids     = world().scene().constitution_tabular().uids();
    auto this_uid = uid();
    if(!std::ranges::binary_search(uids, this_uid))
    {
        throw SimSystemException(fmt::format("Extra Constitution UID ({}) not found in the constitution tabular",
                                             this_uid));
    }

    BuildInfo this_info;
    do_build(this_info);

    m_impl.finite_element_method->add_constitution(this);
}

U64 FiniteElementExtraConstitution::uid() const noexcept
{
    return get_uid();
}

span<const FiniteElementMethod::GeoInfo> FiniteElementExtraConstitution::geo_infos() const noexcept
{
    return m_impl.geo_infos;
}

void FiniteElementExtraConstitution::init()
{
    m_impl.init(uid(), world());

    // let the subclass do the rest of the initialization
    FilteredInfo info{&m_impl};
    do_init(info);
}

void FiniteElementExtraConstitution::do_compute_energy(FiniteElementEnergyProducer::ComputeEnergyInfo& info)
{
    ComputeEnergyInfo this_info{&m_impl.fem(), info.dt(), info.energies()};
    do_compute_energy(this_info);
}

void FiniteElementExtraConstitution::do_compute_gradient_hessian(
    FiniteElementEnergyProducer::ComputeGradientHessianInfo& info)
{
    ComputeGradientHessianInfo this_info{
        &m_impl.fem(), info.dt(), info.gradients(), info.hessians()};
    do_compute_gradient_hessian(this_info);
}

void FiniteElementExtraConstitution::Impl::init(U64 uid, backend::WorldVisitor& world)
{
    using ForEachInfo = FiniteElementMethod::ForEachInfo;

    // 1) Find the geometry slots that have the extra constitution uids containing the given uid
    auto& fem_geo_infos = finite_element_method->m_impl.geo_infos;
    auto  geo_slots     = world.scene().geometries();

    list<SizeT> geo_slot_indices;

    finite_element_method->for_each(
        geo_slots,
        [&](const ForEachInfo& I, geometry::SimplicialComplex& sc)
        {
            auto geoI = I.global_index();
            auto uids = sc.meta().find<VectorXu64>(builtin::extra_constitution_uids);
            if(uids)
            {
                auto extra_uids = uids->view().front();
                for(auto extra_uid : extra_uids)
                {
                    if(extra_uid == uid)
                    {
                        geo_slot_indices.push_back(geoI);
                        // logger::info("Extra constitution {} found in geometry slot {}", uid, I);
                        break;
                    }
                }
            }
        });

    geo_infos.resize(geo_slot_indices.size());

    for(auto&& [i, geo_slot_index] : enumerate(geo_slot_indices))
    {
        geo_infos[i] = fem_geo_infos[geo_slot_index];
    }
}

Float FiniteElementExtraConstitution::BaseInfo::dt() const noexcept
{
    return m_dt;
}

muda::CBufferView<Vector3> FiniteElementExtraConstitution::BaseInfo::xs() const noexcept
{
    return m_impl->xs.view();
}

muda::CBufferView<Vector3> FiniteElementExtraConstitution::BaseInfo::x_bars() const noexcept
{
    return m_impl->x_bars.view();
}

muda::CBufferView<Float> FiniteElementExtraConstitution::BaseInfo::thicknesses() const noexcept
{
    return m_impl->thicknesses.view();
}

span<const FiniteElementMethod::GeoInfo> FiniteElementExtraConstitution::FilteredInfo::geo_infos() const noexcept
{
    return m_impl->geo_infos;
}

span<const Vector3> FiniteElementExtraConstitution::FilteredInfo::positions() noexcept
{
    return m_impl->fem().h_positions;
}

span<const Vector3> FiniteElementExtraConstitution::FilteredInfo::rest_positions() noexcept
{
    return m_impl->fem().h_rest_positions;
}

span<const Float> FiniteElementExtraConstitution::FilteredInfo::thicknesses() noexcept
{
    return m_impl->fem().h_thicknesses;
}
}  // namespace uipc::backend::cuda