#include <finite_element/finite_element_diff_parm_reporter.h>
#include <finite_element/finite_element_method.h>

namespace uipc::backend::cuda
{
void FiniteElementDiffParmReporter::do_build(DiffParmReporter::BuildInfo& info)
{
    m_impl.fem   = &require<FiniteElementMethod>();
    auto dt_attr = world().scene().config().find<Float>("dt");
    m_impl.dt    = dt_attr->view()[0];
    BuildInfo this_info;
    do_build(this_info);
}

void FiniteElementDiffParmReporter::do_assemble(GlobalDiffSimManager::DiffParmInfo& info)
{
    DiffParmInfo this_info{&m_impl, info, m_impl.dt};
    do_assemble(this_info);
}

SizeT FiniteElementDiffParmReporter::DiffParmInfo::frame() const
{
    return m_global_info.frame();
}

IndexT FiniteElementDiffParmReporter::DiffParmInfo::dof_offset(SizeT frame) const
{
    // Frame Dof Offset + FEM Dof Offset => Frame FEM Dof Offset
    return m_global_info.dof_offset(frame) + m_impl->fem->dof_offset(frame);
}

IndexT FiniteElementDiffParmReporter::DiffParmInfo::dof_count(SizeT frame) const
{
    // FEM Dof Count => Frame FEM Dof Count
    return m_impl->fem->dof_count(frame);
}

muda::TripletMatrixView<Float, 1> FiniteElementDiffParmReporter::DiffParmInfo::pGpP() const
{
    IndexT IF = IndexT{frame()};

    auto pGpP = m_global_info.pGpP();

    auto row_offset = dof_offset(IF);
    auto col_offset = 0;

    auto row_count = dof_count(IF);
    auto col_count = pGpP.extent().y;

    return pGpP.submatrix({row_offset, col_offset}, {row_count, col_count});
}

Float FiniteElementDiffParmReporter::DiffParmInfo::dt() const
{
    return m_impl->dt;
}
}  // namespace uipc::backend::cuda
