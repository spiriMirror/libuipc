#include "hip/hip_runtime.h"
#include <kernel_cout.h>
#include <muda/muda.h>
#include <muda/debug.h>
#include <uipc/common/log.h>

namespace uipc::backend::cuda
{
KernelCout::KernelCout()
{
    init();
}

void KernelCout::init()
{
    using namespace muda;

    Debug::set_sync_callback(
        [this]
        {
            m_string_stream.str("");
            m_logger.retrieve(m_string_stream);
            if(m_string_stream.str().empty())
                return;

            log::info(R"( 
-------------------------------------------------------------------------------
*                               Kernel  Console                               *
-------------------------------------------------------------------------------
{}
-------------------------------------------------------------------------------)",
                         m_string_stream.str());
        });
}

muda::LoggerViewer KernelCout::viewer()
{
    // don't delete, just let it go with the program
    // or CUDA may crash at the end of the program
    thread_local static KernelCout* s_instance = nullptr;

    if(!s_instance)
    {
        s_instance = new KernelCout();
        s_instance->init();
    }

    return s_instance->_viewer();
}

muda::LoggerViewer KernelCout::_viewer()
{
    return m_logger.viewer();
}
}  // namespace uipc::backend::cuda